#include "hip/hip_runtime.h"
// Copyright 2025 Ingo Wald
// SPDX-License-Identifier: Apache-2.0

#include "barney/MPIContext.h"
#include "barney/globalTrace/TwoStage.h"
#include "barney/DeviceGroup.h"
#include "barney/render/RayQueue.h"
#include "rtcore/ComputeInterface.h"

namespace BARNEY_NS {

  extern void (*profHook)();
  
  std::vector<std::tuple<double,double,int,const char *>> kernelTimes;

#define TWO_STAGE_PROFILE 0
  
#if TWO_STAGE_PROFILE
# define ENTER() const double prof_t0 = getCurrentTime();
# define LEAVE(count,name)                                    \
  const double prof_t1 = getCurrentTime();                              \
  kernelTimes.push_back(std::make_tuple<double,double,int,const char *>((double)prof_t0,(double)prof_t1,(int)count,(const char *)name));
#else
# define ENTER() /* nothing */
# define LEAVE(count,name) /* nothing */
#endif
  
  int prof_rank;

  void twoStageProfHook()
  {
    std::stringstream ss;
    static double t00 = std::get<0>(kernelTimes[0]);
    for (auto kernel : kernelTimes) {
      double t0 = std::get<0>(kernel)-t00;
      double t1 = std::get<1>(kernel)-t00;
      int numItems = std::get<2>(kernel);
      const char *name = std::get<3>(kernel);

      ss << "r" << prof_rank << " [" << prettyDouble(t0) << "s.."
         << prettyDouble(t1) << "s = "
         << prettyDouble(t1-t0) << "s]: "
         << prettyNumber(numItems) << " items in "
         << name
         << " -> " << prettyDouble(1000000.f*(t1-t0)/numItems) << "s per mio items"
         << std::endl;
    }
    kernelTimes.clear();
    std::cout << ss.str();
  }
  
  __rtc_global
  void buildHitsOnly(const rtc::ComputeInterface &ci,
                      HitOnly *hitOnly,
                      Ray *rayQueue,
                     int N);
  
  __rtc_global
  void reduceReceivedHitsKernel_intraNode(const rtc::ComputeInterface &ci,
                                          HitOnly *hitOnly,
                                          int nRays,
                                          int reduceFactor)
  {
    int tid = ci.launchIndex().x;
    if (tid >= nRays) return;

    HitOnly reduced = hitOnly[tid];
    for (int peer=1;peer<reduceFactor;peer++) {
      HitOnly *hit = hitOnly+peer*nRays+tid;
      
      if (hit->tHit >= reduced.tHit) continue;
      
      reduced = *hit;
    }
    hitOnly[tid] = reduced;
    
  }
  
  __rtc_global
  void reduceReceivedHitsKernel_crossNodes(const rtc::ComputeInterface &ci,
                                           Ray *rayQueueThisRank,
                                           HitOnly *hitOnlyAllRanks,
                                           int nRays,
                                           int reduceFactor)
  {
    int tid = ci.launchIndex().x;
    if (tid >= nRays) return;
    
    Ray ray = rayQueueThisRank[tid];
    for (int peer=0;peer<reduceFactor;peer++) {
      HitOnly *hit = hitOnlyAllRanks+peer*nRays+tid;

      if (hit->tHit >= ray.tMax) continue;

      ray.tMax     = hit->tHit;
      ray.bsdfType = hit->bsdfType;
      ray.hitBSDF  =  hit->hitBSDF;
      ray.P        =  hit->P;
      ray.N        =  hit->N;
    }
    rayQueueThisRank[tid] = ray;
  }
  
  
  __rtc_global
  void createRayOnly(const rtc::ComputeInterface &ci,
                     RayOnly *rayOnly,
                     Ray *rayQueue,
                     int N);
  __rtc_global
  void buildStagedRayQueue(const rtc::ComputeInterface &ci,
                           Ray *rayQueue,
                           RayOnly *rayOnly,
                           int N);
  
  TwoStage::TwoStage(MPIContext *context)
    : GlobalTraceImpl(context),
      context(context),
      world(context->world),
      topo(context->topo.get()),
      logTopo(FromEnv::get()->logTopo),
      logQueues(FromEnv::get()->logQueues),
      opt_mpi(FromEnv::enabled("opt_mpi"))
  {
    prof_rank = world.rank;
#if TWO_STAGE_PROFILE
    profHook = twoStageProfHook;
#endif

    if (context->devices->size() != 1)
      throw std::runtime_error
        ("twostage all2all currently only works for one device per rank");
    this->device = context->devices->get(0);
    
    if (topo->islands.size() != 1)
      throw std::runtime_error
        ("twostage all2all currently only works for a single island");

    myGID = device->globalRank();
    numGlobal = topo->allDevices.size();
    global.rayCounts.resize(numGlobal);
    // sanity check that all physical nodes have same number of GPUs
    std::map<size_t,int> gpuCountInHost;
    int numHosts = 0;
    for (int gid=0; gid<context->topo->allDevices.size(); gid++) {
      auto &dev = context->topo->allDevices[gid];
      gpuCountInHost[dev.hostNameHash]++;
      numHosts = std::max(numHosts,topo->physicalHostIndexOf[gid]+1);
    }
    gpusPerHost = gpuCountInHost.begin()->second;
    for (auto count : gpuCountInHost)
      if (count.second != gpusPerHost)
        throw std::runtime_error
          ("twostage all2all currently requires same number of GPUs on all ranks");
    assert(numHosts * gpusPerHost == context->topo->allDevices.size());

    this->hostIdx = topo->physicalHostIndexOf[myGID];
    // allows oversubscription - we enumerate based on (host:process)
    // instead of (host.physialGPU)
    this->gpuIdx = topo->rankOnHost[myGID];
    _rankOf.resize(numGlobal);

    std::vector<int> logicalGidOfRank(numGlobal);
    int myLogicalGID = this->hostIdx * gpusPerHost + this->gpuIdx;
    world.allGather(logicalGidOfRank.data(),&myLogicalGID,1);
    for (int r=0;r<numGlobal;r++)
      _rankOf[logicalGidOfRank[r]] = r;
    this->numHosts = numGlobal / gpusPerHost;

    if (opt_mpi) {
      crossNodes.comm = world.split(this->gpuIdx);
      crossNodes.rayCounts.resize(crossNodes.comm.size);
      intraNode.comm = world.split(this->hostIdx);
      intraNode.rayCounts.resize(intraNode.comm.size);
    }
    
    if (logTopo) {
      world.barrier();
      if (context->myRank() == 0) {
        std::cout << "=========== TwoStage All2all ===========" << std::endl;
        std::cout << "- num MPI ranks (w/ one gpu each) " << numGlobal << std::endl;
        std::cout << "- detected num physical hosts " << numHosts << std::endl;
        std::cout << "- detected num (active) GPUs per host " << gpusPerHost << std::endl;
        PRINT(numHosts);
        PRINT(gpusPerHost);
        for (int h=0;h<numHosts;h++)
          for (int g=0;g<gpusPerHost;g++) {
            std::cout << "- gpu on rank " << (rankOf(h,g))
                      << " is logical h" << h << "g" << g << " {"
                      << topo->toString(rankOf(h,g)) << "}" << std::endl;
          }
      }
      world.barrier();
    }
  }


  void TwoStage::ensureAllOurQueuesAreLargeEnough()
  {
    auto rtc = device->rtc;
    size_t ourRequiredQueueSize
      = device->rayQueue->size * numGlobal;
    if (ourRequiredQueueSize > currentReservedSize) {
      if (logQueues) {
        std::cout << "resizing ray queues from " << currentReservedSize
                  << " to " << ourRequiredQueueSize << std::endl;
      }
      for (int i=0;i<2;i++)
        if (raysOnly[i]) rtc->freeMem(raysOnly[i]);
      for (int i=0;i<2;i++)
        if (hitsOnly[i]) rtc->freeMem(hitsOnly[i]);
      
      if (stagedRayQueue) rtc->freeMem(stagedRayQueue);
      
      size_t N = ourRequiredQueueSize+1024;
      for (int i=0;i<2;i++)
        raysOnly[i] = (RayOnly*)rtc->allocMem(N*sizeof(RayOnly));
      for (int i=0;i<2;i++)
        hitsOnly[i] = (HitOnly*)rtc->allocMem(N*sizeof(HitOnly));
      stagedRayQueue = (Ray *)rtc->allocMem(N*sizeof(Ray));
      
      currentReservedSize = N;
    }
  }

  // step 1: have all ranks exchange which (global) device has how
  // many rays (needed to set up the send/receives)
  void TwoStage::exchangeHowManyRaysEachDeviceHas()
  {
    ENTER();

    if (opt_mpi) {
    } else {
      int myRayCount = device->rayQueue->numActive;
      world.allGather(global.rayCounts.data(),&myRayCount,1);
    
      if (logQueues)  {
        if (myGID == 0) {
          std::cout << "ray counts (" << global.rayCounts.size() << "):";
          for (auto rc : global.rayCounts) std::cout << " " << rc;
          std::cout << std::endl;
        }
      }
    }
    LEAVE(1,"exchangeHowManyRaysEachDeviceHas");
  }
  
  
  /*! in this stage we have all each GPU exchange its rays with
    all GPUs that have same phsycail ID in all OTHER ranks, but NOT
    with other GPUs in same rank
  */
  void TwoStage::sendAndReceiveRays_crossNodes()
  {
    ENTER();
    // -----------------------------------------------------------------------------
    // first, create 'raysOnly[]' array, for each local device
    // -----------------------------------------------------------------------------
    int myRayCount = device->rayQueue->numActive;
    {
      SetActiveGPU forDuration(device);
      int bs = 128;
      int nb = divRoundUp(myRayCount,bs);
      __rtc_launch(device->rtc,
                   createRayOnly,
                   nb,bs,
                   // args
                   raysOnly[0],
                   device->rayQueue->traceAndShadeReadQueue.rays,
                   myRayCount);
    }


    if (opt_mpi) {
      crossNodes.comm.allGather(crossNodes.rayCounts.data(),&myRayCount,1);

      void *sendBuf = raysOnly[0];
      int sendCount = myRayCount*sizeof(RayOnly);
      void *recvBuf = raysOnly[1];
      std::vector<int> recvCounts(crossNodes.comm.size);
      std::vector<int> recvOffsets(crossNodes.comm.size);
      int sumCounts = 0;
      for (int i=0;i<crossNodes.comm.size;i++) {
        recvOffsets[i] = sumCounts;
        recvCounts[i] = crossNodes.rayCounts[i]*sizeof(RayOnly);
        sumCounts += recvCounts[i];
      }
      crossNodes.sumRaysReceived = sumCounts / sizeof(RayOnly);
      if (logQueues) 
        printf("xchg-rays-cross r%i we have %i sumrecv %i first two counts %i %i\n",
               myGID,myRayCount,crossNodes.sumRaysReceived,
               crossNodes.rayCounts[0],
               crossNodes.rayCounts[1]);
      
      device->rtc->sync();
      BN_MPI_CALL(Allgatherv(sendBuf,sendCount,
                             MPI_BYTE,
                             recvBuf,(int*)recvCounts.data(),(int*)recvOffsets.data(),
                             MPI_BYTE,
                             crossNodes.comm));
    } else {
      device->rtc->sync();
      std::vector<MPI_Request> requests;
      int recvOfs = 0;
      for (int h=0;h<numHosts;h++) {
        MPI_Request req;
        int recvCount = global.rayCounts[rankOf(h,gpuIdx)];
        if (logQueues) 
          printf("splat-cross r%i receiving %i from %i (q 0->1)\n",
                 myGID,recvCount,rankOf(h,gpuIdx));
        world.recv(rankOf(h,gpuIdx),0,raysOnly[1]+recvOfs,
                   recvCount,req);
        recvOfs += recvCount;
        requests.push_back(req);
      }
      crossNodes.sumRaysReceived = recvOfs;
      if (logQueues) 
        printf("splat-cross r%i total received %i\n",
               myGID,crossNodes.sumRaysReceived);
    
      for (int h=0;h<numHosts;h++) {
        MPI_Request req;
        if (logQueues) 
          printf("splat-cross r%i sending %i to %i (q 0->1)\n",
                 myGID,myRayCount,rankOf(h,gpuIdx));
        world.send(rankOf(h,gpuIdx),0,raysOnly[0],myRayCount,req);
        requests.push_back(req);
      }
    
      BN_MPI_CALL(Waitall(requests.size(),requests.data(),MPI_STATUSES_IGNORE));
    }
    LEAVE(recvOfs,"sendAndReceiveRays_crossNodes");
  }
  

  /*! in this stage we have all each GPU exchange its rays with
    all GPUs that have same phsycail ID in all OTHER ranks, but NOT
    with other GPUs in same rank
  */
  void TwoStage::sendAndReceiveRays_intraNode()
  {
    ENTER();

    if (opt_mpi) {
      int numRaysWeHave = crossNodes.sumRaysReceived;
      intraNode.comm.allGather(intraNode.rayCounts.data(),&numRaysWeHave,1);

      void *sendBuf = raysOnly[1];
      void *recvBuf = raysOnly[0];

      int sendCount = numRaysWeHave*sizeof(RayOnly);
      std::vector<int> recvCounts(intraNode.comm.size);
      std::vector<int> recvOffsets(intraNode.comm.size);
      int sumCounts = 0;
      for (int i=0;i<intraNode.comm.size;i++) {
        recvOffsets[i] = sumCounts;
        recvCounts[i] = intraNode.rayCounts[i]*sizeof(RayOnly);
        sumCounts += recvCounts[i];
      }
      intraNode.sumRaysReceived = sumCounts / sizeof(RayOnly);

      if (logQueues) 
        printf("xchg-rays-intra r%i we have %i sumrecv %i\n",
               myGID,numRaysWeHave,intraNode.sumRaysReceived);
      device->rtc->sync();
      BN_MPI_CALL(Allgatherv(sendBuf,sendCount,
                             MPI_BYTE,
                             recvBuf,(int*)recvCounts.data(),(int*)recvOffsets.data(),
                             MPI_BYTE,
                             intraNode.comm));
    } else {
      std::vector<MPI_Request> requests;
      int recvOfs = 0;
      for (int g=0;g<gpusPerHost;g++) {
        MPI_Request req;
        int raysOnPeer = 0;
        for (int h=0;h<numHosts;h++)
          raysOnPeer += global.rayCounts[rankOf(h,g)];
        if (logQueues) 
          printf("splat-intra r%i receiving %i from %i (q 1->0)\n",
                 myGID,raysOnPeer,rankOf(hostIdx,g));
        world.recv(rankOf(hostIdx,g),0,
                   raysOnly[0]+recvOfs,raysOnPeer,
                   req);
        recvOfs += raysOnPeer;
        requests.push_back(req);
      }
      intraNode.sumRaysReceived = recvOfs;
      if (logQueues) 
        printf("splat-intra r%i total received %i\n",
               myGID,intraNode.sumRaysReceived);

      int numRaysWeHave = 0;
      for (int h=0;h<numHosts;h++)
        numRaysWeHave += global.rayCounts[rankOf(h,gpuIdx)];
      for (int g=0;g<gpusPerHost;g++) {
        MPI_Request req;
        if (logQueues) 
          printf("splat-intra r%i sending %i to %i (q 1->0)\n",
                 myGID,numRaysWeHave,rankOf(hostIdx,g));
        world.send(rankOf(hostIdx,g),0,
                   raysOnly[1],numRaysWeHave,
                   req);
        requests.push_back(req);
      }
      BN_MPI_CALL(Waitall(requests.size(),requests.data(),MPI_STATUSES_IGNORE));
    }
    
    LEAVE(recvOfs,"sendAndReceiveRays_intraNode");
  }

  

  void TwoStage::traceRays(GlobalModel *model,
                           uint32_t rngSeed,
                           bool needHitIDs) 
  {
    assert(needHitIDs == false); // not implemented right now
    ensureAllOurQueuesAreLargeEnough();
    exchangeHowManyRaysEachDeviceHas();
    sendAndReceiveRays_crossNodes();
    sendAndReceiveRays_intraNode();

    traceAllReceivedRays(model,rngSeed,needHitIDs);

    exchangeHits_intraNode();
    reduceHits_intraNode();
    exchangeHits_crossNodes();
    reduceHits_crossNodes();
  }


  void TwoStage::traceAllReceivedRays(GlobalModel *model,
                                      uint32_t rngSeed,
                                      bool needHitIDs)
  {
    
    SetActiveGPU forDuration(device);
    int numRaysWeHaveTotal = intraNode.sumRaysReceived;
    {
      ENTER();
      if (logQueues) 
        printf("buildlocalrays r%i total rays %i (q0)\n",
               myGID,numRaysWeHaveTotal);
      __rtc_launch(device->rtc,
                   buildStagedRayQueue,
                   divRoundUp(numRaysWeHaveTotal,1024),1024,
                   // args
                   stagedRayQueue,
                   raysOnly[0],
                   numRaysWeHaveTotal);
      
      device->rtc->sync();
      LEAVE(numRaysWeHaveTotal,"buildStagedRayQueue");
    }
    
    auto savedOriginalRayCount = device->rayQueue->numActive;
    auto savedOriginalRayQueue = device->rayQueue->traceAndShadeReadQueue.rays;
    device->rayQueue->traceAndShadeReadQueue.rays = stagedRayQueue;
    device->rayQueue->numActive = numRaysWeHaveTotal;

    {
      ENTER()
        if (logQueues) 
          printf("localtrace r%i total rays %i\n",
                 myGID,numRaysWeHaveTotal);
      context->traceRaysLocally(model,rngSeed,needHitIDs);
      device->rtc->sync();
      LEAVE(numRaysWeHaveTotal,"localTrace");
    }
    
    if (logQueues) 
      printf("buildhits r%i total rays %i (q0)\n",
             myGID,numRaysWeHaveTotal);
    {
      ENTER();
      __rtc_launch(device->rtc,
                   buildHitsOnly,
                   divRoundUp(numRaysWeHaveTotal,1024),1024,
                   // args
                   hitsOnly[0],
                   stagedRayQueue,
                   numRaysWeHaveTotal);
      device->rtc->sync();
      LEAVE(numRaysWeHaveTotal,"buildHitsOnly");
    }
    device->rayQueue->numActive = savedOriginalRayCount;
    device->rayQueue->traceAndShadeReadQueue.rays = savedOriginalRayQueue;
  }
  
  void TwoStage::exchangeHits_intraNode()
  {
    ENTER();
    if (opt_mpi) {
      void *sendBuf = hitsOnly[0];
      void *recvBuf = hitsOnly[1];
      std::vector<int> sendOffsets(intraNode.comm.size);
      std::vector<int> sendCounts(intraNode.comm.size);
      std::vector<int> recvOffsets(intraNode.comm.size);
      std::vector<int> recvCounts(intraNode.comm.size);
      int recvSum = 0;
      int sendSum = 0;
      for (int i=0;i<intraNode.comm.size;i++) {
        int recvCount = crossNodes.sumRaysReceived;
        int sendCount = intraNode.rayCounts[i];
        recvOffsets[i] = recvSum*sizeof(HitOnly);
        sendOffsets[i] = sendSum*sizeof(HitOnly);
        recvCounts[i] = recvCount*sizeof(HitOnly);
        sendCounts[i] = sendCount*sizeof(HitOnly);
        sendSum += sendCount;
        recvSum += recvCount;
      }
      BN_MPI_CALL(Alltoallv(sendBuf,
                            (const int*)sendCounts.data(),
                            (const int*)sendOffsets.data(),
                            MPI_BYTE,
                            recvBuf,
                            (const int*)recvCounts.data(),
                            (const int*)recvOffsets.data(),
                            MPI_BYTE,
                            intraNode.comm));
    } else {
      std::vector<MPI_Request> requests;
      int recvOfs = 0;
      for (int g=0;g<gpusPerHost;g++) {
        MPI_Request req;
        int recvCount = 0;
        for (int h=0;h<numHosts;h++)
          recvCount += global.rayCounts[rankOf(h,gpuIdx)];
        world.recv(rankOf(hostIdx,g),0,
                   hitsOnly[1]+recvOfs,recvCount,req);
        if (logQueues) 
          printf("xchg-intra r%i receiving %i from %i (q0->1)\n",
                 myGID,recvCount,rankOf(hostIdx,g));
        requests.push_back(req);
        recvOfs += recvCount;
      }

      // and matching sends
      int sendOfs = 0;
      for (int g=0;g<gpusPerHost;g++) {
        int sendCount = 0;
        for (int h=0;h<numHosts;h++)
          sendCount += global.rayCounts[rankOf(h,g)];
        MPI_Request req;
        world.send(rankOf(hostIdx,g),0,
                   hitsOnly[0]+sendOfs,sendCount,req);
        if (logQueues) 
          printf("xchg-intra r%i sending %i to %i (q0->1)\n",
                 myGID,sendCount,rankOf(hostIdx,g));
        requests.push_back(req);
        sendOfs += sendCount;
      }
      BN_MPI_CALL(Waitall(requests.size(),requests.data(),MPI_STATUSES_IGNORE));
    }
    LEAVE(recvOfs,"exchangeHits_intraNode");
  }


  void TwoStage::reduceHits_intraNode()
  {
    ENTER();
    SetActiveGPU forDuration(device);
    int numUniqueRaysThisGPU = crossNodes.sumRaysReceived;
    // int g = gpuIdx;
    // for (int h=0;h<numHosts;h++)
    //   numUniqueRaysThisGPU += rayCounts[rankOf(h,g)];

    if (logQueues) 
      printf("r%i intra-reducing %i sets of %i hits (q1)\n",
             myGID,
             gpusPerHost,
             numUniqueRaysThisGPU);
    __rtc_launch(device->rtc,
                 reduceReceivedHitsKernel_intraNode,
                 divRoundUp(numUniqueRaysThisGPU,128),128,
                 // args
                 hitsOnly[1],
                 numUniqueRaysThisGPU,
                 gpusPerHost);
    device->rtc->sync();
    LEAVE(numUniqueRaysThisGPU*gpusPerHost,"reduceHits_intraNode");
  }
  
  void TwoStage::exchangeHits_crossNodes()
  {
    ENTER();
    if (opt_mpi) {
      int myRayCount = device->rayQueue->numActive;
      void *sendBuf = hitsOnly[1];
      void *recvBuf = hitsOnly[0];
      std::vector<int> sendOffsets(crossNodes.comm.size);
      std::vector<int> sendCounts(crossNodes.comm.size);
      std::vector<int> recvOffsets(crossNodes.comm.size);
      std::vector<int> recvCounts(crossNodes.comm.size);
      int recvSum = 0;
      int sendSum = 0;
      for (int i=0;i<crossNodes.comm.size;i++) {
        int recvCount = myRayCount;
        int sendCount = crossNodes.rayCounts[i];
        recvOffsets[i] = recvSum*sizeof(HitOnly);
        sendOffsets[i] = sendSum*sizeof(HitOnly);
        recvCounts[i] = recvCount*sizeof(HitOnly);
        sendCounts[i] = sendCount*sizeof(HitOnly);
        sendSum += sendCount;
        recvSum += recvCount;
      }
      if (logQueues) 
        printf("xchg-hits-cross r%i myRayCount %i sendSum %i recvSum %i\n",
               myGID,myRayCount,sendSum,recvSum);
      BN_MPI_CALL(Alltoallv(sendBuf,
                            (const int*)sendCounts.data(),
                            (const int*)sendOffsets.data(),
                            MPI_BYTE,
                            recvBuf,
                            (const int*)recvCounts.data(),
                            (const int*)recvOffsets.data(),
                            MPI_BYTE,
                            crossNodes.comm));
    } else {
      std::vector<MPI_Request> requests;
      int recvOfs = 0;
      int recvCount = global.rayCounts[rankOf(hostIdx,gpuIdx)];
      for (int h=0;h<numHosts;h++) {
        MPI_Request req;

        if (logQueues) 
          printf("xchg-intra r%i receiving %i from %i (q1->0)\n",
                 myGID,recvCount,rankOf(h,gpuIdx));
        world.recv(rankOf(h,gpuIdx),0,
                   hitsOnly[0]+recvOfs,recvCount,req);
        requests.push_back(req);
        recvOfs += recvCount;
      }

      // and matching sends
      int sendOfs = 0;
      for (int h=0;h<numHosts;h++) {
        MPI_Request req;
        int sendCount = global.rayCounts[rankOf(h,gpuIdx)];
        if (logQueues) 
          printf("xchg-intra r%i sending %i to %i (q1->0)\n",
                 myGID,sendCount,rankOf(h,gpuIdx));
        world.send(rankOf(h,gpuIdx),0,
                   hitsOnly[1]+sendOfs,sendCount,req);
        requests.push_back(req);
        sendOfs += sendCount;
      }
    
      BN_MPI_CALL(Waitall(requests.size(),requests.data(),MPI_STATUSES_IGNORE));
    }
    LEAVE(recvOfs,"exchangeHits_crossNodes");
  }
  
  void TwoStage::reduceHits_crossNodes()
  {
    ENTER();
    SetActiveGPU forDuration(device);
    if (opt_mpi) {
      int numUniqueRaysThisGPU = device->rayQueue->numActive;
      if (logQueues) 
        printf("r%i cross-reducing %i sets of %i hits (q0)\n",
               myGID,numHosts,numUniqueRaysThisGPU);
      __rtc_launch(device->rtc,
                   reduceReceivedHitsKernel_crossNodes,
                   divRoundUp(numUniqueRaysThisGPU,128),128,
                   // args
                   device->rayQueue->traceAndShadeReadQueue.rays,
                   hitsOnly[0],
                   numUniqueRaysThisGPU,
                   numHosts);
    } else {
      int numUniqueRaysThisGPU = global.rayCounts[rankOf(hostIdx,gpuIdx)];
      if (logQueues) 
        printf("r%i cross-reducing %i sets of %i hits (q0)\n",
               myGID,numHosts,numUniqueRaysThisGPU);
      __rtc_launch(device->rtc,
                 reduceReceivedHitsKernel_crossNodes,
                 divRoundUp(numUniqueRaysThisGPU,128),128,
                 // args
                 device->rayQueue->traceAndShadeReadQueue.rays,
                 hitsOnly[0],
                 numUniqueRaysThisGPU,
                 numHosts);
    }
    device->rtc->sync();
    LEAVE(numUniqueRaysThisGPU*numHosts,"reduceHits_crossNodes");
  }
  
}
