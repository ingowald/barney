#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2025 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/fb/FrameBuffer.h"
#include "barney/fb/TiledFB.h"
#include "barney/render/World.h"
#include "barney/render/DG.h"
#include "barney/render/Renderer.h"
#include "barney/GlobalModel.h"
#include "barney/render/RayQueue.h"
#include "rtcore/TraceInterface.h"

namespace BARNEY_NS {
  namespace render {

#define SCI_VIS_MODE 1
    
#define MAX_DIFFUSE_BOUNCES 1
    
#define ENV_LIGHT_SAMPLING 1

#define USE_MIS 1


#define CLAMP_F_R 13.f


#if RTC_DEVICE_CODE
    inline __rtc_device float square(float f) { return f*f; }
  
    
    enum { MAX_PATH_DEPTH = 10 };

    inline __rtc_device
    float safe_eps(float f, vec3f v)
    {
      // return 1e-4f;
      return max(f,1e-6f*reduce_max(abs(v)));
    }

    
    inline __rtc_device
    bool sampleAreaLights(Light::Sample &ls,
                          const render::World::DD &world,
                          const vec3f P,
                          const vec3f N,
                          Random &random,
                          bool dbg)
    {
      if (world.numQuadLights == 0) return false;
      static const int RESERVOIR_SIZE = 8;
      int   lID[RESERVOIR_SIZE];
      float u[RESERVOIR_SIZE];
      float v[RESERVOIR_SIZE];
      float weights[RESERVOIR_SIZE];
      float sumWeights = 0.f;
      QuadLight::DD light;
      for (int i=0;i<RESERVOIR_SIZE;i++) {
        lID[i] = min(int(random()*world.numQuadLights),
                     world.numQuadLights-1);
        weights[i] = 0.f;
        light = world.quadLights[lID[i]];
        u[i] = random();
        v[i] = random();
        float lightArea = light.area;
#ifndef NDEBUG
        if (lightArea < 0.f)
          printf("INVALID NEGATIVE LIGHT AREA on light %i/%i : %f\n",
                 lID[i],world.numQuadLights,lightArea);
#endif
        vec3f LN = light.normal;
        vec3f LP = light.corner + u[i]*light.edge0 + v[i]*light.edge1;
        vec3f lightDir = LP - P;
        float lightDist = length(lightDir);
        if (lightDist < 1e-3f) continue;
      
        lightDir *= 1.f/lightDist;

        float weight = dot(lightDir,N);
        if (weight <= 1e-3f) continue;
        weight *= -dot(lightDir,LN);
        if (weight <= 1e-3f) continue;
#ifndef NDEBUG
        if (lightArea == 0.f || reduce_max(light.emission) == 0)
          printf("invalid light! %f : %f %f %f\n",
                 lightArea,
                 light.emission.x,
                 light.emission.y,
                 light.emission.z);
#endif
        weight *= (1.f/(lightDist*lightDist)) * lightArea * reduce_max(light.emission);
#ifndef NDEBUG
        if (isnan(sumWeights) || weight < 0.f)
          printf("area lights: weight[%i:%i] is nan or negative: dist  %f area %f emission %f %f %f\n",
                 i,lID[i],lightDist,lightArea,
                 light.emission.x,
                 light.emission.y,
                 light.emission.z);
#endif
        sumWeights += weight;
        weights[i] = weight;
      }
#ifndef NDEBUG
      if (isnan(sumWeights))
        printf("area lights: sumWeights is nan!\n");
#endif
      if (sumWeights == 0.f) return false;
      float r = random()*sumWeights;
      int i=0;
      while (i<RESERVOIR_SIZE && r >= weights[i]) { r-= weights[i]; ++i; }
      if (i == RESERVOIR_SIZE) return false;
    
      light = world.quadLights[lID[i]];
      vec3f LP = light.corner + u[i]*light.edge0 + v[i]*light.edge1;
      vec3f LD = LP-P;
      ls.direction
        = normalize(LD);
      ls.distance
        = length(LD);
      ls.radiance
        = light.emission
        * (light.area * -dot(light.normal,ls.direction)
           / square(ls.distance));
      ls.pdf
        = weights[i]/sumWeights
        * (float(RESERVOIR_SIZE)/float(world.numQuadLights));
#ifndef NDEBUG
      if (ls.pdf <= 0.f)
        printf("invalid area light PDF %f from i %i weight %f sum %f\n",
               ls.pdf,i,weights[i],sumWeights);
#endif
      return true;
    }

    inline __rtc_device
    bool sampleDirLights(Light::Sample &ls,
                         const World::DD &world,
                         const Renderer::DD &renderer,
                         const vec3f P,
                         const vec3f N,
                         Random &random,
                         bool dbg)
    {
      if (world.numDirLights == 0) return false;
      static const int RESERVOIR_SIZE = 2;
      int   lID[RESERVOIR_SIZE];
      float weights[RESERVOIR_SIZE];
      float sumWeights = 0.f;
      DirLight::DD light;
    
      for (int i=0;i<RESERVOIR_SIZE;i++) {
        lID[i] = min(int(random()*world.numDirLights),
                     world.numDirLights-1);
        weights[i] = 0.f;
        light = world.dirLights[lID[i]];
        vec3f light_radiance
          = light.color
          * light.radiance;
        
        vec3f lightDir = -light.direction;
        float weight = dot(lightDir,N);
        if (dbg) printf("light #%i, dir %f %f %f weight %f\n",lID[i],lightDir.x,lightDir.y,lightDir.z,weight);
        if (weight <= 1e-3f) continue;
        weight *= reduce_max(light_radiance);
        if (weight <= 1e-3f) continue;
        weights[i] = weight;
        sumWeights += weight;
      }
      if (sumWeights == 0.f) return false;
      float r = random()*sumWeights;
      int i=0;
      while (i<RESERVOIR_SIZE && r >= weights[i]) { r-= weights[i]; ++i; }
      if (i == RESERVOIR_SIZE) return false;
    
      light = world.dirLights[lID[i]];
      ls.direction
        = -light.direction;
      ls.distance
        = BARNEY_INF;
      ls.radiance
        = light.color*light.radiance;
      ls.pdf
        = weights[i]/sumWeights
        * (float(RESERVOIR_SIZE)/float(world.numDirLights));
      return weights[i] != 0.f;
    }

    inline __rtc_device
    bool sampleEnvLight(Light::Sample &ls,
                        const World::DD &world,
                        const Renderer::DD &renderer,
                        const vec3f P,
                        const vec3f N,
                        Random &random,
                        bool dbg)
    {
      /* in barney, the environment is either a explicit hdri map (in
         EnvMapLight); or a uniform brightness of 'renderer.ambientRadiance' */
      if (world.envMapLight.texture)
        ls = world.envMapLight.sample(random,dbg);
      else {
#if 0
        ls.direction = randomDirection(random);
        ls.radiance  = renderer.ambientRadiance;
        if (dot(ls.direction,N) < 0.f) ls.direction = -ls.direction;
        ls.pdf       = ONE_OVER_TWO_PI;
        ls.distance  = BARNEY_INF;
#else
        ls.direction = randomDirection(random);
        ls.radiance  = renderer.ambientRadiance;
        ls.pdf       = ONE_OVER_FOUR_PI;
        ls.distance  = BARNEY_INF;
#endif
      }
      return true;
    }

    inline __rtc_device
    bool sampleLights(Light::Sample &ls,
                      const World::DD &world,
                      const Renderer::DD &renderer,
                      const vec3f P,
                      const vec3f Ng,
                      Random &random,
#if USE_MIS
                      bool &lightNeedsMIS,
                      bool &lightIsDirLight,
#endif
                      bool dbg)
    {
#if USE_MIS
# if 0
      // huh ... not sure this is correct; setting this to true means
      // we'll always compute MIS weights for shadow and bounce ray as
      // if there was only an env-map light; even though we may
      // acutally have sampled a dir-light. that _may_ be true because
      // even if we did sample a dirlight there still _is_ a pdf for
      // the env-map light... but it's a bit iffy.
      lightNeedsMIS = true;
# else
      lightNeedsMIS = false;
# endif
#endif

#if ENV_LIGHT_SAMPLING
      Light::Sample els;
      float elsWeight
        = (sampleEnvLight(els,world,renderer,P,Ng,random,dbg)
           ? (reduce_max(els.radiance)/els.pdf)
           : 0.f);
      if (dbg)
        printf("els rad %f %f %f pdf %f\n",
               els.radiance.x,
               els.radiance.y,
               els.radiance.z,
               els.pdf);
      // = world.envMapLight.sample(random,dbg);
#else
      float elsWeight = 0.f;
#endif

      Light::Sample als;
      float alsWeight
        = (sampleAreaLights(als,world,P,Ng,random,dbg)
           ? (reduce_max(als.radiance)/als.pdf)
           : 0.f);
      Light::Sample dls;
      float dlsWeight
        = (sampleDirLights(dls,world,renderer,P,Ng,random,dbg)
           ? (reduce_max(dls.radiance)/dls.pdf)
           : 0.f);

      if (dbg) printf("sampling lights dls %f els %f\n",
                      dlsWeight,elsWeight);
      
      float sumWeights
        = alsWeight+dlsWeight+elsWeight;
      if (sumWeights == 0.f) return false;

      elsWeight *= 1.f/sumWeights;
      alsWeight *= 1.f/sumWeights;
      dlsWeight *= 1.f/sumWeights;
      
      float r = random();
      if (dbg) printf(" light sample %f in cdf %f %f %f\n",
                      r,alsWeight,elsWeight,dlsWeight);
      if (r <= alsWeight) {
        ls = als;
        ls.pdf *= alsWeight;
#if ENV_LIGHT_SAMPLING
      } else if (r <= alsWeight+elsWeight) {
        ls = els;
        ls.pdf *= elsWeight;
        if (dbg) printf(" ->  picked env light sample\n");
# if USE_MIS
        lightNeedsMIS = true;
# endif
#endif
      } else {
        ls = dls;
        ls.pdf *= dlsWeight;
# if USE_MIS
        lightIsDirLight = true;
# endif
        if (dbg) printf(" ->  picked DIR light sample, dls weight %f pdf %f\n",dlsWeight,ls.pdf);
      }
      if (isnan(ls.pdf) || (ls.pdf <= 0.f)) return false;
      
      return true;
    }




    inline __rtc_device
    float schlick(float cosine,
                  float ref_idx)
    {
      float r0 = (1.0f - ref_idx) / (1.0f + ref_idx);
      r0 = r0 * r0;
      return r0 + (1.0f - r0)*powf((1.0f - cosine), 5.0f);
    }
  
  

    inline __rtc_device
    bool refract(const vec3f& v,
                 const vec3f& n,
                 float ni_over_nt,
                 vec3f &refracted)
    {
      vec3f uv = normalize(v);
      float dt = dot(uv, n);
      float discriminant = 1.0f - ni_over_nt * ni_over_nt*(1 - dt * dt);
      if (discriminant > 0.f) {
        refracted = ni_over_nt * (uv - n * dt) - n * sqrtf(discriminant);
        return true;
      }
      else
        return false;
    }
  
    inline __rtc_device
    vec3f radianceFromEnv(const World::DD &world,
                          const Renderer::DD &renderer,
                          Ray &ray)
    {
      auto &env = world.envMapLight;
      if (env.texture) {
        vec3f d = xfmVector(env.toLocal,normalize(ray.dir));
        float theta = pbrtSphericalTheta(d);
        float phi   = pbrtSphericalPhi(d);
        const float invPi  = 1.f/(float)M_PI;
        const float inv2Pi = 1.f/(2.f* (float)M_PI);
        vec2f uv(phi * inv2Pi, theta * invPi);

        vec4f color = rtc::tex2D<vec4f>(env.texture,uv.x,uv.y);
        return env.scale*vec3f(color.x,color.y,color.z);
      } else {
        return renderer.ambientRadiance;
      }
    }

    /*! if there _is_ a dedicated env-map light specified, this looks
      up the background color from that map; otherwise, it returns
      the 'ray.misscolor' that the primary ray generation has set as
      default color for this ray */
    inline __rtc_device
    vec3f primaryRayMissColor(const World::DD &world,
                              const Renderer::DD &renderer,
                              Ray &ray)
    {
      if (world.envMapLight.texture)
        return radianceFromEnv(world,renderer,ray);
      return
        // primary rays do store a default misscolor in the ray itself
        // - we simply return this if there's no env-map.
        (const vec3f&)ray.missColor;
    }

    /*! ugh - that should all go into material::AnariPhysical .... */
    inline __rtc_device
    void bounce(int rayID,
                const World::DD &world,
                const Renderer::DD &renderer,
                vec3f &fragment,
                Ray &ray,
                PathState &state,
                Ray &shadowRay,
                PathState &shadowState,
                int pathDepth)
    {
      
      const float EPS = 1e-4f;

      const bool  hadNoIntersection  = !ray.hadHit();
      const vec3f incomingThroughput = state.throughput;
      
      bool dbg = ray.dbg();
      if (dbg)
        printf("(%i) ------------------------------------------------------------------\n -> incoming %f %f %f dir %f %f %f t %f\n  tp %f %f %f ismiss %i, bsdf %i\n",
               pathDepth,
               ray.org.x,
               ray.org.y,
               ray.org.z,
               (float)ray.dir.x,
               (float)ray.dir.y,
               (float)ray.dir.z,
               ray.tMax,
               (float)state.throughput.x,
               (float)state.throughput.y,
               (float)state.throughput.z,
               int(hadNoIntersection),(int)ray.bsdfType);
      
      if (ray.isShadowRay) {
        // ==================================================================
        // shadow ray = all we have to do is add carried radiance if it
        // reached the light, and discards
        // ==================================================================
                 
        if (hadNoIntersection) {
          fragment =
# if USE_MIS
            (float)state.misWeight *
#endif
            (vec3f)state.throughput;
          if (dbg)
            printf("_shadow_ ray reaches light: tp %f %f %f misweight %f frag %f %f %f\n",
                   (float)state.throughput.x,
                   (float)state.throughput.y,
                   (float)state.throughput.z,
                   (float)state.misWeight,
                   fragment.x,
                   fragment.y,
                   fragment.z);
          if (dbg) printf("shadow miss, frag %f %f %f\n",
                          fragment.x,
                          fragment.y,
                          fragment.z);
        }

        // this path is done.
        shadowRay.tMax = -1.f;
        ray.tMax = -1.f;
        return;
      }

      vec3f Ng = ray.getN();
      if (dbg) printf("ray.N %f %f %f\n",Ng.x,Ng.y,Ng.z);
      const bool  isVolumeHit        = (Ng == vec3f(0.f));
      if (!isVolumeHit)
        Ng = normalize(Ng);
      const bool  hitWasOnFront      = dot((vec3f)ray.dir,Ng) < 0.f;
      vec3f Ngff
        = hitWasOnFront
        ?   Ng
        : - Ng;

      if (hadNoIntersection) {
        // ==================================================================
        // regular ray that did NOT hit ANYTHING 
        // ==================================================================
        if (pathDepth == 0) {
          // ----------------------------------------------------------------
          // PRIMARY ray that didn't hit anything -> background
          // ----------------------------------------------------------------
          fragment = primaryRayMissColor(world,renderer,ray);

          if (dbg)
            printf("miss primary %f %f %f -> %f %f %f\n",
                   ray.missColor.x,
                   ray.missColor.y,
                   ray.missColor.z,
                   fragment.x,fragment.y,fragment.z);
        } else {
          // ----------------------------------------------------------------
          // SECONDARY ray that didn't hit anything -> env-light
          // ----------------------------------------------------------------
          // this path had at least one bounce, but now bounced into
          // nothingness - compute env-light contribution, and weigh it
          // with the path's carried throughput.
#if ENV_LIGHT_SAMPLING
# if USE_MIS
          const vec3f fromEnv = radianceFromEnv(world,renderer,ray);
          fragment = (vec3f)state.throughput * fromEnv * (float)state.misWeight;

          if (dbg)
            printf("bounce ray hits env light: tp %f %f %f misweight %f fromEnv %f %f %f\n",
                   (float)state.throughput.x,
                   (float)state.throughput.y,
                   (float)state.throughput.z,
                   (float)state.misWeight,
                   fromEnv.x,
                   fromEnv.y,
                   fromEnv.z);
# else
          fragment
            = ray.isSpecular
            ? radianceFromEnv(world,renderer,path)
            : vec3f(0.f);
# endif
#else
          const vec3f fromEnv = radianceFromEnv(world,renderer,ray);
          if (dbg)
            printf("fromenv %f %f %f\n",
                   fromEnv.x,
                   fromEnv.y,
                   fromEnv.z);
          fragment = state.throughput * fromEnv;
#endif
        }
        // no outgoing rays; this path is done.
        ray.tMax = -1.f;
        return;
      }
    

      // ==================================================================
      // this ray DID hit something: compute its local frame buffer
      // contribution at this hit point (if any), and generate secondary
      // ray and shadow ray (if applicable), with proper weights.
      // ==================================================================    
      Random random(ray.rngSeed.next((const uint32_t&)ray.tMax));//rayID,ray.rngSeed);
      // Random random(ray.rngSeed.next((const uint32_t&)ray.tMax));//rayID,ray.rngSeed);
      const PackedBSDF bsdf = ray.getBSDF();
      // bool doTransmission = false;
      // =  ((float)ray.mini.transmission > 0.f)
      // && (random() < (float)ray.mini.transmission);
      render::DG dg;
      dg.P  = ray.P;
      dg.Ng = Ng;
      dg.Ns = Ng;
      dg.wo = -normalize((vec3f)ray.dir);
      dg.insideMedium = ray.isInMedium;

      // if the ray is a volume hit we want it offset it into the
      // direction the ray came from (otherwise we have a chance of
      // the shadow ray or boucne ray to terminate right where the
      // original ray ended; for others we want to offset based on
      // normal */
      const float offsetEpsilon = safe_eps(EPS,dg.P);
      vec3f frontFacingSurfaceOffset
        = (isVolumeHit?dg.wo:Ngff);

      // ==================================================================
      // FIRST, let us look at generating any shadow rays, if
      // applicable; this way we can later modify the incoming ray in
      // place when we generate the outgoing ray.
      // ==================================================================
      Light::Sample ls;
      // todo check if BSDF is perfectly specular
#if USE_MIS
      bool lightNeedsMIS = false;
      bool lightIsDirLight = false;
#endif
      if (dbg)
        printf("sampling lights with N %f %f %f\n",Ngff.x,Ngff.y,Ngff.z);
      if (sampleLights(ls,world,renderer,dg.P,Ngff,random,
#if USE_MIS
                       lightNeedsMIS,
                       lightIsDirLight,
#endif
                       dbg)) {
        if (dbg)
          printf("sample light dir %f %f %f rad %f %f %f pdf %f spike %f\n",
                 ls.direction.x,
                 ls.direction.y,
                 ls.direction.z,
                 ls.radiance.x,
                 ls.radiance.y,
                 ls.radiance.z,
                 ls.pdf,
                 reduce_max(ls.radiance)/ls.pdf);
        EvalRes f_r
          = bsdf.eval(dg,ls.direction,dbg);
        if (dbg) printf("eval light res %f %f %f: %f\n",
                        f_r.value.x,
                        f_r.value.y,
                        f_r.value.z,
                        f_r.pdf);
          
        if (!f_r.valid() || reduce_max(f_r.value) < 1e-4f) {
          if (dbg)
            printf(" no f_r, killing shadow ray\n");
          shadowRay.tMax = -1.f;
        } else {
#ifdef CLAMP_F_R
          f_r.value = min(f_r.value,vec3f(CLAMP_F_R));
#endif
          vec3f tp_sr
            = (incomingThroughput)
            * (1.f/ls.pdf)
            * f_r.value
            * ls.radiance
            // * ONE_OVER_PI
            * (isVolumeHit?1.f:fabsf(dot(dg.Ng,ls.direction)))
            ;

          
          if (dbg) {
            printf(" -> inc tp %f %f %f, dot %f\n",
                   incomingThroughput.x,
                   incomingThroughput.y,
                   incomingThroughput.z,
                   (isVolumeHit?1.f:fabsf(dot(dg.Ng,ls.direction))));
            printf(" -> shadow f_r %f %f %f ls.rad %f %f %f pdf %f\n",
                   f_r.value.x,
                   f_r.value.y,
                   f_r.value.z,
                   ls.radiance.x,
                   ls.radiance.y,
                   ls.radiance.z,
                   ls.pdf);
            printf(" -> shadow ray tp %f %f %f\n",
                   tp_sr.x,
                   tp_sr.y,
                   tp_sr.z);
          }
          makeShadowRay
            (shadowRay,shadowState,
             /* thrghhpt */tp_sr,
             /* surface: */dg.P + offsetEpsilon*frontFacingSurfaceOffset,
             /* to light */ls.direction,
             /* length   */ls.distance * (1.f-2.f*offsetEpsilon));
          shadowRay.rngSeed = ray.rngSeed;// + 1; random();
          ray.rngSeed.next((const uint32_t&)ray.tMax);
          // Random rng(ray.rngSeed.next(hash(ti.getRTCInstanceIndex(),
          //                        ti.getGeometryIndex(),
          //                        ti.getPrimitiveIndex())));

          shadowRay._dbg = ray._dbg;
          shadowState.pixelID = state.pixelID;
            
          shadowState.misWeight = 1.f;
#if USE_MIS
          if (!lightIsDirLight && lightNeedsMIS) {
            float pdf_lightRay_lightDir
              = world.envMapLight.pdf(ls.direction);
            float pdf_scatterRay_lightDir
              = bsdf.pdf(dg,ls.direction);
            // compute MIS weight weight that shadow direction
            shadowState.misWeight
              = pdf_lightRay_lightDir
              / (pdf_lightRay_lightDir + pdf_scatterRay_lightDir + 1e-10f);
            // and if it's too small for any reason, kill the shadow
            // ray
            if ((float)shadowState.misWeight < 1e-5f)
              shadowRay.tMax  = -1.f;
          }
#endif
        }
      }
      
      // ==================================================================
      // now, let's decide what to do with the ray itself
      // ==================================================================

      // if we exceeded max depth we die, one way or another.
      if (pathDepth >= MAX_PATH_DEPTH) {
        ray.tMax = -1.f;
        return;
      }
      // now per default, "create" a valid scatter ray.
      ray.clearHit();
      ray.tMax = BARNEY_INF;
      
      ScatterResult scatterResult;
      bsdf.scatter(scatterResult,dg,random,dbg);
#ifndef NDEBUG
      if (scatterResult.type == ScatterResult::INVALID)
        printf("broken BSDF, doesn't set scatter type!\n");
#endif 
      if (dbg)
        printf("scatter result.valid ? %i\n",
               int(scatterResult.valid()));
      if (!scatterResult.valid() || scatterResult.pdf <= 1e-6f)
        return;
      
      if (scatterResult.type == ScatterResult::VOLUME) {
#if SCI_VIS_MODE
        // sci vis mode: volumes do shadow, but nothing more
        ray.tMax = -1.f;
        return;
#else
        // treat volume scatter like a diffuse scatter.
        scatterResult.type = ScatterResult::DIFFUSE;
#endif
      }
      
      if (scatterResult.type == ScatterResult::DIFFUSE ||
          scatterResult.type == ScatterResult::VOLUME) {
        if (state.numDiffuseBounces >= MAX_DIFFUSE_BOUNCES) {
          ray.tMax = -1.f;
          return;
        } else
          state.numDiffuseBounces = state.numDiffuseBounces + 1;
      }
      ray.isSpecular = (scatterResult.type == ScatterResult::SPECULAR);
      
      if (dbg)
        printf("offsetting into sign %f, direction %f %f %f\n",
               scatterResult.offsetDirection,
               frontFacingSurfaceOffset.x,
               frontFacingSurfaceOffset.y,
               frontFacingSurfaceOffset.z); 
      ray.org
        = dg.P + scatterResult.offsetDirection * offsetEpsilon*frontFacingSurfaceOffset;
      // #ifdef CLAMP_F_R
      //       scatterResult.f_r = min(scatterResult.f_r,vec3f(100.f));
      // #endif

      if (dbg)
        printf("path scattered, bsdf in scatter dir is %f %f %f, pdf %f\n",
               (float)scatterResult.f_r.x, 
               (float)scatterResult.f_r.y, 
               (float)scatterResult.f_r.z,
               scatterResult.pdf);
      ray.dir
        = normalize(scatterResult.dir);
      
      vec3f scatterFactor
        = scatterResult.f_r
        * (isVolumeHit?1.f:fabsf(dot(dg.Ng,normalize(ray.dir))))
        // * ONE_OVER_PI
        / (isinf(scatterResult.pdf)? 1.f : (//ONE_PI*
                                            scatterResult.pdf + 1e-10f));
      
#if 1
      // uhhhh.... this is TOTALLY wrong, but let's limit how much
      // each bounce can increase the throughput of a ray ... this
      // makes fireflies go away (well, makes them go 'less', but can
      // lose a lot of envergy if the brdf sample code isn't close to
      // the actual brdf.
      scatterFactor = min(scatterFactor,vec3f(2.5f));
#endif

      state.throughput
        = state.throughput * scatterFactor;
      if (dbg && scatterResult.changedMedium)
        printf("path DID change medium\n");
      if (scatterResult.changedMedium)
        ray.isInMedium = !ray.isInMedium;
      
      if (dbg)
        printf("scatter dir %f %f %f tp %f %f %f\n",
               (float)ray.dir.x,
               (float)ray.dir.y,
               (float)ray.dir.z,
               (float)state.throughput.x,
               (float)state.throughput.y,
               (float)state.throughput.z);
      
      
#if USE_MIS
      if (lightNeedsMIS && !isinf(scatterResult.pdf)) {
        float pdf_scatterRay_scatterDir = bsdf.pdf(dg,ray.dir);
        float pdf_lightRay_scatterDir   = world.envMapLight.pdf(ray.dir);
        
        state.misWeight
          = pdf_scatterRay_scatterDir
          / (pdf_scatterRay_scatterDir + pdf_lightRay_scatterDir);
      } else {
        state.misWeight = 1.f;
      }
#endif
    }
#endif // device code  

    //     struct PathTraceKernel {
    // #ifdef RTC_DEVICE_CODE
    //       inline __rtc_device
    //       void run(const rtc::ComputeInterface &rt);
    // #endif
      
    //       World::DD world;
    //       Renderer::DD renderer;
    //       AccumTile *accumTiles;
    //       AuxTiles   auxTiles;
    //       int accumID;
    //       SingleQueue readQueue;
    //       int numRays;
    //       SingleQueue writeQueue;
    //       int *d_nextWritePos;
    //       int generation;
    //     };

#if RTC_DEVICE_CODE
    // inline __rtc_device
    // void PathTraceKernel::run
    __rtc_global void _shadeRays(const rtc::ComputeInterface &rt,
                                 World::DD world,
                                 Renderer::DD renderer,
                                 AccumTile *accumTiles,
                                 AuxTiles   auxTiles,
                                 int accumID,
                                 SingleQueue readQueue,
                                 int numRays,
                                 SingleQueue writeQueue,
                                 int *d_nextWritePos,
                                 int generation
                                 )
    {
      int tid = rt.getThreadIdx().x + rt.getBlockIdx().x*rt.getBlockDim().x;
      if (tid >= numRays) return;

      Ray ray = readQueue.rays[tid];
      PathState state = readQueue.states[tid];
#ifdef NDEBUG
      enum { dbg = false };
#else
      bool dbg = ray.dbg();
#endif
      /* note(iw): IMHO pixels that did _not_ hit any geometry should
         have an alpha value of 0, even if they did comptue a 'color'
         from the env map. However, TSD blends its renderd image over
         a black background, in which case that blends away background
         pixels, so for now turn that off. The "right" fix for this
         would be for TSD to handle alpha properly. */
#define COMPUTE_PROPER_ALPHA_CHANNEL 0
      float alpha 
        = (generation == 0)
        ?
#if COMPUTE_PROPER_ALPHA_CHANNEL
        (ray.hadHit()? 1.f : ray.missColor.w)
#else
        1.f
#endif
        : 0.f;
      float incomingZ = ray.tMax;
      vec3f incomingN
        = ray.hadHit()
        ? ray.getN()
        : vec3f(0.f);
      // if (incomingN == vec3f(0.f))
      //   incomingN = vec3f(1.f,0.f,0.f);
      // what we'll add into the frame buffer
      vec3f fragment = 0.f;
      // create a (potential) shadow ray, and init to 'invalid'
      Ray shadowRay;
      PathState shadowState;
      shadowRay.tMax = -1.f;
      
      // bounce that ray on the scene, possibly generating a) a fragment
      // to add to frame buffer; b) a outgoing ray (in-place
      // modification of 'path'); and/or c) a shadow ray
      bounce(tid,
             world,renderer,
             fragment,
             ray,state,
             shadowRay,shadowState,
             generation);
    
      // if (ray.crosshair && !dbg) {
      //   fragment = vec3f(1.f,0.f,0.f);
      // }
      
      // write shadow and bounce ray(s), if any were generated
      if (dbg)
        printf("ray.tmax %f shadowray.tmax %f frag %f %f %f\n",
               ray.tMax,shadowRay.tMax,
               fragment.x,fragment.y,fragment.z);
      if (shadowRay.tMax > 0.f) {
        int pos = rt.atomicAdd(d_nextWritePos,1);
        writeQueue.rays[pos] = shadowRay;
        writeQueue.states[pos] = shadowState;
      }
      if (ray.tMax > 0.f) {
        int pos = rt.atomicAdd(d_nextWritePos,1);
        writeQueue.rays[pos] = ray;
        writeQueue.states[pos] = state;
      }

      // and write the shade fragment, if generated
      int tileID  = int(state.pixelID / pixelsPerTile);
      int tileOfs = int(state.pixelID % pixelsPerTile);
      vec4f &valueToAccumInto
        = accumTiles[tileID].accum[tileOfs];

#if DENOISE
      vec3f &valueToAccumNormalInto
        = accumTiles[tileID].normal[tileOfs];
#endif
      
      // ==================================================================
      // add to accum buffer. be careful of two things:
      //
      // a) since each pixel could have two DIFFERENT rays in the
      // queue (shadow ray and bounce ray) we cannot simply 'add', but
      // have to use an atomic add, because these could be in the same
      // warp.
      //
      // b) since we don't have an explicit frame buffer clear we
      // still have to make sure each pixel is written - not added -
      // exactly once in the first generation of the first frame.
      // ==================================================================

      // clamping ...
      float clampMax = 10.f*(1+accumID);
      fragment = min(fragment,vec3f(clampMax));

      if (accumID == 0 && generation == 0) {
        valueToAccumInto = vec4f(fragment.x,fragment.y,
                                 fragment.z,alpha);

        // write aux buffers (depth, normal, hitIDs
        accumTiles[tileID].normal[tileOfs] = incomingN;
        if (auxTiles.depth) 
          auxTiles.depth[tileID] . f[tileOfs] = incomingZ;
        if (auxTiles.primID)
          auxTiles.primID[tileID].ui[tileOfs] = readQueue.hitIDs[tid].primID;
        if (auxTiles.objID)
          auxTiles.objID[tileID] .ui[tileOfs] = readQueue.hitIDs[tid].objID;
        if (auxTiles.instID)
          auxTiles.instID[tileID].ui[tileOfs] = readQueue.hitIDs[tid].instID;
        
      } else {
        // we're either an accumluated frame, or a non-primary bounce
        // of the first frame; either way we'll accumulate color and
        // ignore anything else.
        if (generation == 0 && alpha > 0.f) 
          rt.atomicAdd(&valueToAccumInto.w,alpha);

        // printf("frag %f %f %f:%f\n",
        //        fragment.x,fragment.y,fragment.z,alpha);
        if (fragment.x > 0.f)
          rt.atomicAdd(&valueToAccumInto.x,fragment.x);
        if (fragment.y > 0.f)
          rt.atomicAdd(&valueToAccumInto.y,fragment.y);
        if (fragment.z > 0.f)
          rt.atomicAdd(&valueToAccumInto.z,fragment.z);
      }
    }
#endif
  }  

  using namespace render;
  
  void Context::shadeRaysLocally(Renderer *renderer,
                                 GlobalModel *model,
                                 FrameBuffer *fb,
                                 int generation,
                                 uint32_t rngSeed)
  {
    for (auto slotModel : model->modelSlots) {
      World *world = slotModel->world.get();
      for (auto device : *world->devices) {
        SetActiveGPU forDuration(device);
        RayQueue *rayQueue = device->rayQueue;
        device->rayQueue->resetWriteQueue();
        
        TiledFB *devFB     = fb->getFor(device);
        int numRays        = rayQueue->numActive;
        if (numRays == 0) continue;
        int bs = 128;
        int nb = divRoundUp(numRays,bs);
        World::DD devWorld
          = world->getDD(device// ,rngSeed
                         );
        Renderer::DD devRenderer
          = renderer->getDD(device);

        // render::PathTraceKernel args = {
        //   devWorld,devRenderer,
        //   devFB->accumTiles,
        //   devFB->auxTiles,
        //   (int)fb->accumID,
        //   rayQueue->traceAndShadeReadQueue,
        //   numRays,
        //   rayQueue->receiveAndShadeWriteQueue,
        //   rayQueue->_d_nextWritePos,
        //   generation,
        // };
        if (FromEnv::get()->logQueues) {
          std::stringstream ss;
          ss << "#bn" << myRank() << ": ## ray queue kernel SHADE " << std::endl
             << "  from " << rayQueue->traceAndShadeReadQueue.rays
             << " + " << rayQueue->traceAndShadeReadQueue.states << std::endl
             << "  to   " << rayQueue->receiveAndShadeWriteQueue.rays
             << " + " << rayQueue->receiveAndShadeWriteQueue.states << std::endl;
          std::cout << ss.str();
        }
        
          
        // device->shadeRays->launch(nb,bs,&args);
        __rtc_launch(//device
                     device->rtc,
                     //kernel
                     _shadeRays,
                     //config
                     nb,bs,
                     //args
                     devWorld,devRenderer,
                     devFB->accumTiles,
                     devFB->auxTiles,
                     (int)fb->accumID,
                     rayQueue->traceAndShadeReadQueue,
                     numRays,
                     rayQueue->receiveAndShadeWriteQueue,
                     rayQueue->_d_nextWritePos,
                     generation
                     );
      }
    }

    // ------------------------------------------------------------------
    // wait for kernel to complete, and swap queues
    // ------------------------------------------------------------------
    for (auto device : *devices) {
      SetActiveGPU forDuration(device);
      device->rtc->sync();
      device->rayQueue->swapAfterShade();
      device->rayQueue->numActive = device->rayQueue->readNumActive();
    }
  }
  
}

