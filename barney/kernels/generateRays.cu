#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/DeviceContext.h"
#include "barney/Ray.h"
#include "barney/fb/FrameBuffer.h"

namespace barney {

  /*! generates a new wave-front of rays, to be written to
      'rayQueue[]', at (atomically incrementable) positoin
      *d_count. This kernel operates on *tiles* (not complete frames);
      the list of tiles to generate rays for is passed in 'tileDescs';
      there will be one cuda block per tile */
  __global__
  void g_generateRays(/*! the camera used for generating the rays */
                      Camera::DD camera,
                      /*! a unique random number seed value for pixel
                          and lens jitter; probably just accumID */
                      int rngSeed,
                      int accumID,
                      /*! full frame buffer size, to check if a given
                          tile's pixel ID is still valid */
                      vec2i fbSize,
                      /*! pointer to a device-side int that tracks the
                          next write position in the 'write' ray
                          queue; can be atomically incremented on the
                          device */
                      int *d_count,
                      /*! pointer to device-side ray queue to write
                          newly generated raysinto */
                      Ray *rayQueue,
                      /*! tile descriptors for the tiles that the
                          frame buffer owns on this device; rays
                          should only get generated for these tiles */
                      TileDesc *tileDescs)
  {
    __shared__ int l_count;
    if (threadIdx.x == 0)
      l_count = 0;

    // ------------------------------------------------------------------
    __syncthreads();
    
    int tileID = blockIdx.x;
    
    vec2i tileOffset = tileDescs[tileID].lower;
    int ix = (threadIdx.x % tileSize) + tileOffset.x;
    int iy = (threadIdx.x / tileSize) + tileOffset.y;

    Ray ray;
    ray.pixelID = tileID * (tileSize*tileSize) + threadIdx.x;
    Random rand(ix+fbSize.x*accumID,
                iy+fbSize.y*accumID);
    // Random rand(rngSeed,ray.pixelID);
    // rand();
    // rand();
    // rand();
    
    ray.org  = camera.lens_00;
    ray.dir
      = camera.dir_00
      + ((ix+((accumID==0)?.5f:rand()))/float(fbSize.x))*camera.dir_du
      + ((iy+((accumID==0)?.5f:rand()))/float(fbSize.y))*camera.dir_dv;
    ray.dir = normalize(ray.dir);

    bool centerPixel = ((ix == fbSize.x/2) && (iy == fbSize.y/2));
    ray.dbg         = centerPixel;
    ray.hadHit      = false;
    ray.isShadowRay = false;
    ray.isInMedium  = false;
    ray.rngSeed     = rand.state;
    ray.tMax        = 1e30f;

    // if (ray.dbg)
    //   printf("-------------------------------------------------------\n");
    // if (ray.dbg)
    //   printf("  # generating INTO %lx\n",rayQueue);
             
    // if (ray.dbg)
    //   printf("spawned %f %f %f dir %f %f %f\n",
    //          ray.org.x,
    //          ray.org.y,
    //          ray.org.z,
    //          (float)ray.dir.x,
    //          (float)ray.dir.y,
    //          (float)ray.dir.z);

    const float t = (iy+.5f)/float(fbSize.y);
    // for *primary* rays we pre-initialize basecolor to a background
    // color; this way the shaderays function doesn't have to reverse
    // engineer pixel pos etc
    
#if 0    
    ray.hit.baseColor = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
    ray.hit.baseColor = .5f*ray.hit.baseColor*ray.hit.baseColor;
#else
    ray.hit.baseColor = vec3f(0.0f, 0.0f, 0.0f);
#endif

#if 0    
    bool crossHair = ((ix == fbSize.x/2) || (iy == fbSize.y/2));
    if (crossHair && !ray.dbg)
      ray.hit.baseColor = vec3f(1,0,0);
#endif      
    
    ray.hit.N = vec3f(0.f);
    ray.throughput = vec3f(1.f);
    
    int pos = -1;
    if (ix < fbSize.x && iy < fbSize.y) 
      pos = atomicAdd(&l_count,1);

    // ------------------------------------------------------------------
    __syncthreads();
    if (threadIdx.x == 0) 
      l_count = atomicAdd(d_count,l_count);
    
    // ------------------------------------------------------------------
    __syncthreads();
    if (pos >= 0) 
      rayQueue[l_count + pos] = ray;
  }
  
  void DeviceContext::generateRays_launch(TiledFB *fb,
                                          const Camera::DD &camera,
                                          int rngSeed)
  {
    auto device = fb->device;
    SetActiveGPU forDuration(device);
    
    g_generateRays
      <<<fb->numActiveTiles,pixelsPerTile,0,device->launchStream>>>
      (camera,
       rngSeed,
       fb->owner->accumID,
       fb->numPixels,
       rays._d_nextWritePos,
       rays.receiveAndShadeWriteQueue,
       fb->tileDescs);
  }
}
