#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/geometry/Cones.h"
#include "rtcore/TraceInterface.h"

RTC_DECLARE_GLOBALS(BARNEY_NS::render::OptixGlobals);

namespace BARNEY_NS {
  using namespace BARNEY_NS::render;

  inline __rtc_device float sqrt(float f) { return sqrtf(f); }
  inline __rtc_device float inversesqrt(float f) { return 1./sqrtf(f); }
  inline __rtc_device float length2(vec3f v) { return dot(v,v); }
  
  struct ConesPrograms {
#if RTC_DEVICE_CODE
    /*! bounding box program */
    static inline __rtc_device
    void bounds(const rtc::TraceInterface &ti,
                const void *geomData,
                owl::common::box3f &bounds,  
                const int32_t primID)
    {
      auto &self = *(Cones::DD*)geomData;
      const vec2i pidx
        = self.indices
        ? self.indices[primID]
        : (2 * primID + vec2i(0, 1));
    
      const vec3f pa = (const vec3f&)self.vertices[pidx.x];
      const vec3f pb = (const vec3f&)self.vertices[pidx.y];
    
      const float ra = self.radii[pidx.x];
      const float rb = self.radii[pidx.y];
      box3f aBox(pa-ra,pa+ra);
      box3f bBox(pb-ra,pb+rb);
    
      bounds.lower = min(aBox.lower,bBox.lower);
      bounds.upper = max(aBox.upper,bBox.upper);
    }

    /*! closest-hit program - doesn't do anything because we do all the
      work in IS prog, but needs to exist to make optix happy */
    static inline __rtc_device
    void closestHit(rtc::TraceInterface &ti)
    {
      /* nothing - already set in isec */
    }
    
    static inline __rtc_device
    void anyHit(rtc::TraceInterface &ti)
    {
      /* nothing - already set in isec */
    }
  
    /*! largely stolen from VisRTX */
    static inline __rtc_device
    void intersect(rtc::TraceInterface &ti)
    {
      Ray &ray    = *(Ray*)ti.getPRD();
      const auto &self
        = *(Cones::DD*)ti.getProgramData();
      const int primID = ti.getPrimitiveIndex();
      const int instID = ti.getInstanceID();
      const OptixGlobals &globals = OptixGlobals::get(ti);
      const World::DD &world = globals.world;
        
      render::HitAttributes hitData;
      hitData.primID          = primID;
      hitData.instID          = instID;
      const DeviceMaterial &material
        = world.materials[self.materialID];
      hitData.t = ray.tMax;
      float ray_tmin = ti.getRayTmin();
      
      vec3f ro  = ti.getObjectRayOrigin();
      vec3f rd  = ti.getObjectRayDirection();
    
      const vec2i idx
        = self.indices
        ? self.indices[primID]
        : (2 * primID + vec2i(0, 1));
    
      const auto p0 = (const vec3f &)self.vertices[idx.x];
      const auto p1 = (const vec3f &)self.vertices[idx.y];

      const float ra = self.radii[idx.x];
      const float rb = self.radii[idx.y];

      const vec3f ba = p1 - p0;
      const vec3f oa = ro - p0;
      const vec3f ob = ro - p1;

      const float m0 = dot(ba, ba);
      const float m1 = dot(oa, ba);
      const float m2 = dot(ob, ba);
      const float m3 = dot(rd, ba);

      float lerp_t = 0.f;

      // interpolator for anari-style color/attribute interpolation
      auto interpolator = [&](const GeometryAttribute::DD &attrib) -> vec4f
      {
        const vec4f value_a = attrib.fromArray.valueAt(idx.x);
        const vec4f value_b = attrib.fromArray.valueAt(idx.y);
        const vec4f ret = (1.f-lerp_t)*value_a + lerp_t*value_b;
        // printf("lerp (%i) %f %f %f and (%i) %f %f %f f %f\n",
        //        idx.x,
        //        value_a.x,
        //        value_a.y,
        //        value_a.z,
        //        idx.y,
        //        value_b.x,
        //        value_b.y,
        //        value_b.z,
        //        lerp_t);
        return ret;
      };

      if (m1 < 0.0f) {
        if (length2(oa * m3 - rd * m1) < (ra * ra * m3 * m3)) {
          float t = -m1 / m3;
          if (t > ray_tmin && t < hitData.t) {
            lerp_t = 0.f;
            vec3f N = normalize(-ba * inversesqrt(m0));
            vec3f P = (vec3f)ro+t*rd;

            hitData.t               = t;
            hitData.objectPosition  = P;
            hitData.objectNormal    = N;
            hitData.worldPosition
              = ti.transformPointFromObjectToWorldSpace(P);
            hitData.worldNormal
              = normalize(ti.transformNormalFromObjectToWorldSpace(N));
          
            // trigger the anari attribute evaluation
            self.setHitAttributes(hitData,interpolator,world,ray.dbg());
          
            // ... store the hit in the ray, rqs-style ...
            material.setHit(ray,hitData,world.samplers,ray.dbg());
          
            // .... and let optix know we did have a hit.
            ti.reportIntersection(hitData.t, 0);
            return;
          }
        }
      } else if (m2 > 0.0f) {
        if (length2(ob * m3 - rd * m2) < (rb * rb * m3 * m3)) {
          lerp_t = 1.f;
          float t = -m2 / m3;
          if (t > ray_tmin && t < hitData.t) {
            vec3f N = normalize(ba * inversesqrt(m0));
            vec3f P = (vec3f)ro+t*rd;

            hitData.t               = t;
            hitData.objectPosition  = P;
            hitData.objectNormal    = N;
            hitData.worldPosition
              = ti.transformPointFromObjectToWorldSpace(P);
            hitData.worldNormal
              = normalize(ti.transformNormalFromObjectToWorldSpace(N));
                    
            // trigger the anari attribute evaluation
            self.setHitAttributes(hitData,interpolator,world,ray.dbg());
          
            // ... store the hit in the ray, rqs-style ...
            material.setHit(ray,hitData,world.samplers,ray.dbg());
          
            // .... and let optix know we did have a hit.
            ti.reportIntersection(hitData.t, 0);
            return;
          }
        }
      }
      
      const float m4 = dot(rd, oa);
      const float m5 = dot(oa, oa);
      const float rr = ra - rb;
      const float hy = m0 + rr * rr;

      float k2 = m0 * m0 - m3 * m3 * hy;
      float k1 = m0 * m0 * m4 - m1 * m3 * hy + m0 * ra * (rr * m3 * 1.0f);
      float k0 = m0 * m0 * m5 - m1 * m1 * hy + m0 * ra * (rr * m1 * 2.0f - m0 * ra);

      const float h = k1 * k1 - k2 * k0;
      if (h < 0.0f)
        return;

      const float t = (-k1 - sqrtf(h)) / k2;
      const float y = m1 + t * m3;
      if (y > 0.0f && y < m0 && t > ray_tmin && t < hitData.t) {
        vec3f N = normalize(m0 * (m0 * (oa + t * rd) + rr * ba * ra) - ba * hy * y);
        vec3f P = (vec3f)ro+t*rd;
        lerp_t = y / m0;
        //lerp_t = dot(P-p1,p0-p1)/dot(p0-p1,p0-p1);
        lerp_t = clamp(lerp_t,0.f,1.f);
        hitData.primID          = primID;
        hitData.t               = t;
        hitData.objectPosition  = P;
        hitData.objectNormal    = N;
        hitData.worldPosition
          = ti.transformPointFromObjectToWorldSpace(P);
        hitData.worldNormal
          = normalize(ti.transformNormalFromObjectToWorldSpace(N));
        
        // trigger the anari attribute evaluation
        self.setHitAttributes(hitData,interpolator,world,ray.dbg());
        
        // ... store the hit in the ray, rqs-style ...
        material.setHit(ray,hitData,world.samplers,ray.dbg());
        
        // .... and let optix know we did have a hit.
        ti.reportIntersection(hitData.t, 0);
      }
    }
#endif
  };
  
  RTC_EXPORT_USER_GEOM(Cones,Cones::DD,ConesPrograms,false,false);
} // ::BARNEY_NS

