#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2025 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/api/Context.h"
#if BARNEY_MPI
# include "barney/common/MPIWrappers.h"
# include "barney/barney_mpi.h"
#endif

static_assert(sizeof(size_t) == 8, "Trying to compile in 32-bit mode ... this isn't going to work");

#define WARN_NOTIMPLEMENTED std::cout << " ## " << __PRETTY_FUNCTION__ << " not implemented yet ..." << std::endl;

#if 0
# define LOG_API_ENTRY std::cout << OWL_TERMINAL_BLUE << "#bn: " << __FUNCTION__ << OWL_TERMINAL_DEFAULT << std::endl;
#else
# define LOG_API_ENTRY /**/ 
#endif

#ifdef NDEBUG 
# define BARNEY_ENTER(fct) /* nothing */
# define BARNEY_LEAVE(fct,retValue) /* nothing */
#else
# define BARNEY_ENTER(fct) try {                                 \
  if (0) std::cout << "@bn.entering " << fct << std::endl;       \
  

# define BARNEY_LEAVE(fct,retValue)                                     \
  } catch (std::exception &e) {                                         \
    std::cerr << OWL_TERMINAL_RED << "@" << fct << ": "                 \
              << e.what() << OWL_TERMINAL_DEFAULT << std::endl;         \
    return retValue ;                                                   \
  }
#endif

namespace barney_api {

  FromEnv::FromEnv()
  {
    const char *e = getenv("BARNEY_CONFIG");
    if (!e) return;
    std::vector<std::string> components;
    std::string es = e;
    while (true) {
      size_t p = es.find(":");
      if (p == es.npos) {
        components.push_back(es);
        break;
        }
      components.push_back(es.substr(0,p));
      es = es.substr(p+1);
    }
    std::map<std::string,std::string> keyValue;
    for (auto comp : components) {
      size_t p = comp.find("=");
      if (p == comp.npos) {
        keyValue[comp] = "";
      } else {
        keyValue[comp.substr(0,p)] = comp.substr(p+1);
      }
    }
    for (auto kv : keyValue) {
      const std::string key = kv.first;
      const std::string value = kv.second;
      
      std::cout << "#barney.config " << key << " = '" << value << "'" << std::endl;

      if (value == "on" || value == "ON" || value == "1")
        boolValues[key] = 1;
      else if (value == "off" || value == "OFF" || value == "0")
        boolValues[key] = 0;
      
      if (key == "LOG_QUEUES" || key == "log_queues")
        logQueues = true;
      else if (key == "SKIP_DENOISING")
        skipDenoising = true;
      else if (key == "LOG_CONFIG" || key == "log_config")
        logConfig = true;
      else if (key == "LOG_BACKEND")
        logBackend = true;
      else if (key == "LOG_TOPO" || key == "log_topo")
        logTopo = true;
      else
        std::cerr << "Warning: unknown or unrecognized BARNEY_CONFIG key '" << key << "'" << std::endl;
    }
  }
  const FromEnv *FromEnv::get()
  {
    static std::mutex mutex;
    std::lock_guard<std::mutex> lock(mutex);
    static FromEnv *singleton = 0;
    if (!singleton) singleton = new FromEnv;
    return singleton;
  }
  
  extern "C" {
#if BARNEY_BACKEND_EMBREE
    barney_api::Context *
    createContext_embree(const std::vector<int> &dgIDs);
#endif
#if BARNEY_BACKEND_OPTIX
    barney_api::Context *
    createContext_optix(const std::vector<int> &dgIDs,
                        int numGPUs, const int *gpuIDs);
#endif
#if BARNEY_BACKEND_CUDA
    barney_api::Context *
    createContext_cuda(const std::vector<int> &dgIDs,
                       int numGPUs, const int *gpuIDs);
#endif
#if BARNEY_MPI
# if BARNEY_BACKEND_EMBREE
    barney_api::Context *
    createMPIContext_embree(barney_api::mpi::Comm world,
                            const std::vector<int> &dgIDs);
# endif
# if BARNEY_BACKEND_OPTIX
    barney_api::Context *
    createMPIContext_optix(barney_api::mpi::Comm world,
                           const std::vector<int> &dgIDs,
                           int numGPUs, const int *gpuIDs);
# endif
# if BARNEY_BACKEND_CUDA
    barney_api::Context *
    createMPIContext_cuda(barney_api::mpi::Comm world,
                           const std::vector<int> &dgIDs,
                           int numGPUs, const int *gpuIDs);
# endif
#endif
  }
  
  inline Context *checkGet(BNContext context)
  {
    assert(context);
    return (Context *)context;
  }
  
  inline Renderer *checkGet(BNRenderer renderer)
  {
    assert(renderer);
    return (Renderer *)renderer;
  }
  
  inline Model *checkGet(BNModel model)
  {
    assert(model);
    return (Model *)model;
  }
  
  inline std::string checkGet(const char *s)
  {
    assert(s != nullptr);
    return s;
  }
  
  inline Object *checkGet(BNObject object)
  {
    if (!object) throw std::runtime_error
                   ("@barney: trying to use/access null object");
    assert(object);
    return (Object *)object;
  }

  inline Data *checkGet(BNData data)
  {
    assert(data);
    return (Data *)data;
  }

  inline Camera *checkGet(BNCamera camera)
  {
    assert(camera);
    return (Camera *)camera;
  }
  
  // inline GlobalModel *checkGet(BNModel model)
  // {
  //   assert(model);
  //   return (GlobalModel *)model;
  // }
  
  // inline ModelSlot *checkGet(BNModel model, int slot)
  // {
  //   return checkGet(model)->getSlot(slot);
  // }
  
  inline Geometry *checkGet(BNGeom geom)
  {
    if (!geom) throw std::runtime_error("@barney: trying to use/access null geometry");
    assert(geom);
    return (Geometry *)geom;
  }
  
  inline Volume *checkGet(BNVolume volume)
  {
    if (!volume) throw std::runtime_error("@barney: trying to use/access null volume");
    assert(volume);
    return (Volume *)volume;
  }
  
  inline ScalarField *checkGet(BNScalarField sf)
  {
    assert(sf);
    return (ScalarField *)sf;
  }
  
  inline Group *checkGet(BNGroup group)
  {
    if (!group) throw std::runtime_error
                   ("@barney: trying to use/access null group");
    assert(group);
    return (Group *)group;
  }
  
  
  inline FrameBuffer *checkGet(BNFrameBuffer frameBuffer)
  {
    assert(frameBuffer);
    return (FrameBuffer *)frameBuffer;
  }

  // ------------------------------------------------------------------
  inline std::shared_ptr<Group> checkGetSP(BNGroup group)
  {
    return checkGet(group)->shared_from_this()->as<Group>();
  }
  
  // ------------------------------------------------------------------
  inline std::shared_ptr<Data> checkGetSP(BNData data)
  {
    return checkGet(data)->shared_from_this()->as<Data>();
  }
  
  inline std::shared_ptr<ScalarField> checkGetSP(BNScalarField sf)
  {
    return checkGet(sf)->shared_from_this()->as<ScalarField>();
  }
  // ------------------------------------------------------------------

  /*! creates a cudaArray2D of specified size and texels. Can be passed
    to a sampler to create a matching cudaTexture2D, or as a background
    image to a renderer */
  BARNEY_API
  BNTextureData bnTextureData2DCreate(BNContext _context,
                                      int slot,
                                      BNDataType texelFormat,
                                      int width, int height,
                                      const void *texels)
  {
    LOG_API_ENTRY;
    Context *context = checkGet(_context);
    
    std::shared_ptr<TextureData> td
      = context->createTextureData(slot,
                                     texelFormat,
                                     vec3i(width,height,0),
                                     texels);
    return (BNTextureData)context->initReference(td);
  }
  
  BARNEY_API
  BNTextureData bnTextureData3DCreate(BNContext _context,
                                      int slot,
                                      BNDataType texelFormat,
                                      int width, int height, int depth,
                                      const void *texels)
  {
    LOG_API_ENTRY;
    Context *context = checkGet(_context);
    std::shared_ptr<TextureData> td
      = context->createTextureData(slot,
                                   texelFormat,
                                   vec3i(width,height,depth),
                                   texels);
    return (BNTextureData)context->initReference(td);
  }

  
  // ------------------------------------------------------------------
  BARNEY_API
  BNTexture2D bnTexture2DCreate(BNContext _context,
                                int slot,
                                BNDataType texelFormat,
                                /*! number of texels in x dimension */
                                uint32_t size_x,
                                /*! number of texels in y dimension */
                                uint32_t size_y,
                                const void *texels,
                                BNTextureFilterMode  filterMode,
                                BNTextureAddressMode addressMode_x,
                                BNTextureAddressMode addressMode_y,
                                BNTextureColorSpace  colorSpace)
  {
    LOG_API_ENTRY;
    Context *context = checkGet(_context);
#if 1
    std::shared_ptr<TextureData> td
      = context->createTextureData(slot,texelFormat,
                                   vec3i(size_x,size_y,0),texels);
    // BNTextureData td
    //   = bnTextureData2DCreate(_context,slot,texelFormat,
    //                           size_x,size_y,texels);
    BNTextureAddressMode addressModes[3] = {
      addressMode_x,addressMode_y,(BNTextureAddressMode)0
    };
    std::shared_ptr<Texture> tex
      = context->createTexture(td,
                               filterMode,
                               addressModes,
                               colorSpace);
    return (BNTexture2D)context->initReference(tex);
#else
    auto devices = context->getDevices(slot);
    TextureData::SP td
      = std::make_shared<TextureData>(context,devices,
                                      texelFormat,
                                      vec3i(size_x,size_y,0),
                                      texels);
    Texture::SP tex
      = std::make_shared<Texture>(context,devices,
                                  td,filterMode,
                                  addressMode_x,addressMode_y,
                                  colorSpace);
    
    return (BNTexture)context->initReference(tex);
#endif
  }

  BARNEY_API
  BNTexture3D bnTexture3DCreate(BNContext _context,
                                int slot,
                                BNDataType texelFormat,
                                uint32_t size_x,
                                uint32_t size_y,
                                uint32_t size_z,
                                const void *texels,
                                BNTextureFilterMode  filterMode,
                                BNTextureAddressMode addressMode)
  {
    LOG_API_ENTRY;
    Context *context = checkGet(_context);
#if 1
    std::shared_ptr<TextureData> td
      = context->createTextureData(slot,texelFormat,
                                   vec3i(size_x,size_y,size_z),texels);
    BNTextureAddressMode addressModes[3] = {
      addressMode,addressMode,addressMode
    };
    std::shared_ptr<Texture> tex
      = context->createTexture(td,
                               filterMode,
                               addressModes,
                               BN_COLOR_SPACE_LINEAR);
    return (BNTexture3D)context->initReference(tex);
#else
    auto devices = context->getDevices(slot);
    TextureData::SP td
      = std::make_shared<TextureData>(context,devices,
                                      texelFormat,
                                      vec3i(size_x,size_y,size_z),
                                      texels);
    Texture3D::SP tex
      = std::make_shared<Texture3D>(context,devices,
                                    td,filterMode,addressMode);
    
    return (BNTexture3D)context->initReference(tex);
#endif
  }
  
  // ------------------------------------------------------------------
  
  BARNEY_API
  BNModel bnModelCreate(BNContext _context)
  {
    LOG_API_ENTRY;
    Context *context = checkGet(_context);
    std::shared_ptr<Model> model = context->createModel();
    return (BNModel)context->initReference(model);
  }

  BARNEY_API
  BNRenderer bnRendererCreate(BNContext _context,
                              const char *ignoreForNow)
  {
    LOG_API_ENTRY;
    Context *context = checkGet(_context);
    std::shared_ptr<Renderer> renderer = context->createRenderer();
    return (BNRenderer)context->initReference(renderer);
  }

  /*! allows for setting one of 5 attribute arrays for the given slot's
    model. */
  BARNEY_API
  void bnSetInstanceAttributes(BNModel model,
                               int slot,
                               const char *whichAttribute,
                               BNData value)
  {
    LOG_API_ENTRY;
    Data::SP data
      = value
      ? ((Data *)value)->shared_from_this()->as<Data>()
      : Data::SP{};
    checkGet(model)->setInstanceAttributes(slot,whichAttribute,data);
  }

  
  BARNEY_API
  void bnSetInstances(BNModel model,
                      int slot,
                      BNGroup *_groups,
                      BNTransform *xfms,
                      int numInstances)
  {
    LOG_API_ENTRY;
    checkGet(model)->setInstances(slot,
                                  (Group **)_groups,
                                  (const affine3f *)xfms,
                                  numInstances);
  }
  
  BARNEY_API
  void  bnRelease(BNObject _object)
  {
    LOG_API_ENTRY;
    Object *object = checkGet(_object);
    assert(object);
    Context *context = object->getContext();
    assert(context);
    context->releaseHostReference(object->shared_from_this());
  }
  
  BARNEY_API
  void  bnAddReference(BNObject _object)
  {
    LOG_API_ENTRY;
    if (_object == 0) return;
    Object *object = checkGet(_object);
    Context *context = object->getContext();
    context->addHostReference(object->shared_from_this());
  }


  BARNEY_API
  void bnContextDestroy(BNContext context)
  {
    LOG_API_ENTRY;
    delete (Context *)context;
  }

  BARNEY_API
  BNScalarField bnScalarFieldCreate(BNContext _context,
                                    int slot,
                                    const char *type)
  {
    Context *context = checkGet(_context);
    std::shared_ptr<ScalarField> sf
      = context->createScalarField(slot,type);
    return (BNScalarField)context->initReference(sf);
  }
  
  
  BARNEY_API
  BNGeom bnGeometryCreate(BNContext _context,
                          int slot,
                          const char *type)
  {
    Context *context = checkGet(_context);
    std::shared_ptr<Geometry> geom
      = context->createGeometry(slot,type);
    return (BNGeom)context->initReference(geom);
  }

  BARNEY_API
  BNMaterial bnMaterialCreate(BNContext _context,
                              int slot,
                              const char *type)
  {
    Context *context = checkGet(_context);
    std::shared_ptr<Material> material
      = context->createMaterial(slot,type);
    return (BNMaterial)context->initReference(material);
  }

  BARNEY_API
  BNSampler bnSamplerCreate(BNContext _context,
                            int slot,
                            const char *type)
  {
    Context *context = checkGet(_context);
    std::shared_ptr<Sampler> sampler
      = context->createSampler(slot,type);
    if (!sampler) return 0;
    return (BNSampler)context->initReference(sampler);
  }
  
  BARNEY_API
  BNCamera bnCameraCreate(BNContext _context,
                          const char *type)
  {
    Context *context = checkGet(_context);
    std::shared_ptr<Camera> camera
      = context->createCamera(type);
    if (!camera) return 0;
    return (BNCamera)context->initReference(camera);
  }

  BARNEY_API
  void bnVolumeSetXF(BNVolume volume,
                     bn_float2 domain,
                     const bn_float4 *_values,
                     int numValues,
                     float densityAt1)
  {
    LOG_API_ENTRY;
    assert(_values);

    checkGet(volume)->setXF(range1f(domain.x,domain.y),
                            _values,numValues,
                            densityAt1);
  }
  
  BARNEY_API
  BNVolume bnVolumeCreate(BNContext _context,
                          int slot,
                          BNScalarField _sf)
  {
    LOG_API_ENTRY;
    Context *context = checkGet(_context);
    
    std::shared_ptr<ScalarField> sf = checkGetSP(_sf);
    std::shared_ptr<Volume> volume
      = context->createVolume(checkGetSP(_sf));
    return (BNVolume)context->initReference(volume);
  }

  BARNEY_API
  BNLight bnLightCreate(BNContext _context,
                        int slot,
                        const char *type)
  {
    LOG_API_ENTRY;
    Context *context = checkGet(_context);
    std::shared_ptr<Light> light
      = context->createLight(slot,type);
    return (BNLight)context->initReference(light);
  }

  BARNEY_API
  BNData bnDataCreate(BNContext _context,
                      int slot,
                      BNDataType dataType,
                      size_t numItems,
                      const void *items)
  {
    LOG_API_ENTRY;
    Context *context = checkGet(_context);
    std::shared_ptr<Data> data
      = context->createData(slot,dataType);
    data->set(items,numItems);
    return (BNData)context->initReference(data);
  }

      BARNEY_API
  void bnDataSet(BNData _data,
                 size_t numItems,
                 const void *items)
  {
    Data::SP data = checkGetSP(_data);
    data->set(items,numItems);
  }

  


  BARNEY_API
  BNGroup bnGroupCreate(BNContext _context,
                        int slot,
                        BNGeom *geoms, int numGeoms,
                        BNVolume *volumes, int numVolumes)
  {
    LOG_API_ENTRY;
    BARNEY_ENTER(__PRETTY_FUNCTION__);
    Context *context = checkGet(_context);
    std::shared_ptr<Group>
      group = context->createGroup(slot,
                                   (Geometry **)geoms,numGeoms,
                                   (Volume **)volumes,numVolumes);
    return (BNGroup)context->initReference(group);
    BARNEY_LEAVE(__PRETTY_FUNCTION__,0);
  }

  BARNEY_API
  void  bnGroupBuild(BNGroup group)
  {
    LOG_API_ENTRY;
    BARNEY_ENTER(__PRETTY_FUNCTION__);
    if (!group) {
#ifndef NDEBUG
      std::cerr << "@barney(WARNING): bnGroupBuild with null group - ignoring this, but this is is an app error that should be fixed, and is only likely to cause issues later on" << std::endl;
#endif
      return;
    }
    checkGet(group)->build();
    BARNEY_LEAVE(__PRETTY_FUNCTION__,);
  }
  
  BARNEY_API
  void  bnBuild(BNModel model,
                int slot)
  {
    BARNEY_ENTER(__PRETTY_FUNCTION__);
    LOG_API_ENTRY;
    checkGet(model)->build(slot);
    BARNEY_LEAVE(__PRETTY_FUNCTION__,);
  }
  
  BARNEY_API
  void bnCommit(BNObject target)
  {
    LOG_API_ENTRY;
    checkGet(target)->commit();
  }
              
  BARNEY_API
  void bnSetString(BNObject target, const char *param, const char *value)
  {
    if (!checkGet(target)->setString(checkGet(param),value))
      checkGet(target)->warn_unsupported_member(param,"std::string");
  }

  BARNEY_API
  void bnSetData(BNObject target, const char *param, BNData value)
  {
    Data::SP data
      = value
      ? checkGetSP(value)
      : Data::SP{};
    if (!checkGet(target)->setData(checkGet(param),data))
      checkGet(target)->warn_unsupported_member(param,"BNData");
  }

  BARNEY_API
  void bnSetObject(BNObject target, const char *param, BNObject value)
  {
    Object::SP asObject
      = value 
      ? checkGet(value)->shared_from_this()
      : Object::SP{};
    bool accepted = checkGet(target)->setObject(checkGet(param),asObject);
    if (!accepted)
      checkGet(target)->warn_unsupported_member(param,"BNObject");
  }

  BARNEY_API
  void bnSet1i(BNObject target, const char *param, int x)
  {
    if (!checkGet(target)->set1i(checkGet(param),x))
      checkGet(target)->warn_unsupported_member(param,"int");
  }

  BARNEY_API
  void bnSet2i(BNObject target, const char *param, int x, int y)
  {
    if (!checkGet(target)->set2i(checkGet(param),vec2i(x,y)))
      checkGet(target)->warn_unsupported_member(param,"vec2i");
  }

  BARNEY_API
  void bnSet3i(BNObject target, const char *param, int x, int y, int z)
  {
    if (!checkGet(target)->set3i(checkGet(param),vec3i(x,y,z)))
      checkGet(target)->warn_unsupported_member(param,"vec3i");
  }

# ifdef __VECTOR_TYPES__
  BARNEY_API
  void bnSet3ic(BNObject target, const char *param, int3 value)
  {
    if (!checkGet(target)->set3i(checkGet(param),(const vec3i&)value))
      checkGet(target)->warn_unsupported_member(param,"vec3i");
  }
#endif
  
  BARNEY_API
  void bnSet4i(BNObject target, const char *param, int x, int y, int z, int w)
  {
    if (!checkGet(target)->set4i(checkGet(param),vec4i(x,y,z,w)))
      checkGet(target)->warn_unsupported_member(param,"vec4i");
  }

  BARNEY_API
  void bnSet1f(BNObject target, const char *param, float value)
  {
    if (!checkGet(target)->set1f(checkGet(param),value))
      checkGet(target)->warn_unsupported_member(param,"float");
  }

  BARNEY_API
  void bnSet3f(BNObject target, const char *param, float x, float y, float z)
  {
    LOG_API_ENTRY;
    if (!checkGet(target)->set3f(checkGet(param),vec3f(x,y,z)))
      checkGet(target)->warn_unsupported_member(param,"vec3f");
  }

  BARNEY_API
  void bnSet4f(BNObject target, const char *param, float x, float y, float z, float w)
  {
    if (!checkGet(target)->set4f(checkGet(param),vec4f(x,y,z,w)))
      checkGet(target)->warn_unsupported_member(param,"vec4f");
  }

  // BARNEY_API
  // void bnSet3fc(BNObject target, const char *param, float3 value)
  // {
  //   if (!checkGet(target)->set3f(checkGet(param),(const vec3f&)value))
  //     checkGet(target)->warn_unsupported_member(param,"vec3f");
  // }

  // BARNEY_API
  // void bnSet4fc(BNObject target, const char *param, float4 value)
  // {
  //   if (!checkGet(target)->set4f(checkGet(param),(const vec4f&)value))
  //     checkGet(target)->warn_unsupported_member(param,"vec4f");
  // }
  
  BARNEY_API
  void bnSet4x3fv(BNObject target, const char *param, const BNTransform *transform)
  {
    assert(transform);
    if (!checkGet(target)->set4x3f(checkGet(param),*(const affine3f*)transform))
      checkGet(target)->warn_unsupported_member(param,"affine3f");
  }

  BARNEY_API
  void bnSet4x4fv(BNObject target, const char *param, const bn_float4 *transform)
  {
    assert(transform);
    if (!checkGet(target)->set4x4f(checkGet(param),(const vec4f*)transform))
      checkGet(target)->warn_unsupported_member(param,"mat4f");
  }
  



  

  
  BARNEY_API
  BNFrameBuffer bnFrameBufferCreate(BNContext _context, int deprecated)
  {
    LOG_API_ENTRY;
    Context *context = checkGet(_context);
    std::shared_ptr<FrameBuffer> fb
      = context->createFrameBuffer();
    return (BNFrameBuffer)context->initReference(fb);
  }

  BARNEY_API
  void bnFrameBufferResize(BNFrameBuffer fb,
                           BNDataType colorFormat,
                           int sizeX, int sizeY,
                           uint32_t channels)
  {
    LOG_API_ENTRY;
    checkGet(fb)->resize(colorFormat,vec2i{sizeX,sizeY},channels);
  }

  BARNEY_API
  void bnFrameBufferRead(BNFrameBuffer fb,
                         BNFrameBufferChannel channel,
                         void *hostPtr,
                         BNDataType requestedFormat)
  {
    LOG_API_ENTRY;
    checkGet(fb)->read(channel,hostPtr,requestedFormat);
  }
  
  BARNEY_API
  void bnAccumReset(BNFrameBuffer fb)
  {
    checkGet(fb)->resetAccumulation();
  }
  
  BARNEY_API
  void bnRender(BNRenderer renderer,
                BNModel    model,
                BNCamera   camera,
                BNFrameBuffer fb)
  {
    // static double t_first = getCurrentTime();
    // static double t_sum = 0.;

    // double t0 = getCurrentTime();
    static int numCalls = 0;
    if (++numCalls < 10)
      LOG_API_ENTRY;
    checkGet(model)->render(checkGet(renderer),checkGet(camera),checkGet(fb));
    // double t1 = getCurrentTime();

    // t_sum += (t1-t0);
    // printf("time in %f\n",float((t_sum / (t1 - t_first))));
  }

  BARNEY_API
  BNContext bnContextCreate(/*! how many data slots this context is to
                              offer, and which part(s) of the
                              distributed model data these slot(s)
                              will hold */
                            const int *dataRanksOnThisContext,
                            int        numDataRanksOnThisContext,
                            /*! which gpu(s) to use for this
                              process. default is to distribute
                              node's GPUs equally over all ranks on
                              that given node */
                            const int *_gpuIDs,
                            int  numGPUs)
  {
    LOG_API_ENTRY;
    if (getenv("BARNEY_FORCE_CPU")) {
      if (FromEnv::get()->logBackend) {
        std::cout << "#bn. found BARNEY_FORCE_CPU flag." << std::endl;
      }
      static int negOne = -1;
      _gpuIDs = &negOne;
      numGPUs = 1;
    }

    if (FromEnv::get()->logBackend) {
      std::cout << "#bn. creating context over numGPUs = " << numGPUs << " gpu IDs ";
      if (_gpuIDs == nullptr)
        std::cout << "<null>" << std::endl;
      else {
        for (int i=0;i<numGPUs;i++)
          std::cout << _gpuIDs[i] << " ";
        std::cout << std::endl;
      }
    }
    
    
    try {
      // ------------------------------------------------------------------
      // create vector of data groups; if actual specified by user we
      // use those; otherwise we use IDs
      // [0,1,...numModelSlotsOnThisHost)
      // ------------------------------------------------------------------
      assert(numDataRanksOnThisContext > 0);
      std::vector<int> dataGroupIDs;
      for (int i = 0;i < numDataRanksOnThisContext;i++)
        dataGroupIDs.push_back
          (dataRanksOnThisContext
           ? dataRanksOnThisContext[i]
           : i);

      // ------------------------------------------------------------------
      // create a backend. logic is as follows:
      //
      // 1) user can _explicitly_ request a CPU device by asking for a
      // single GPU with ID=-1 (ie, numGPUs=1,gpuIDs={-1}). If so,
      // create a CPU device if possible.
      // ------------------------------------------------------------------
      if (
#if BARNEY_BACKEND_EMBREE && !BARNEY_BACKEND_OPTIX
          1
#else
          numGPUs == 1 && _gpuIDs[0] == -1          
#endif
          ) {
# if BARNEY_BACKEND_EMBREE
        return (BNContext)createContext_embree(dataGroupIDs);
# else
        throw std::runtime_error
          ("explicitly asked for CPU backend, "
           "but cpu/embree backend not compiled in");
# endif
      }

      // ------------------------------------------------------------------
      // 2) if user did specify a list of GPUs, create a GPU backend,
      // or return an error.
      // ------------------------------------------------------------------
      if (_gpuIDs != nullptr) {
#if BARNEY_BACKEND_OPTIX
        return (BNContext)createContext_optix(dataGroupIDs,numGPUs,_gpuIDs);
#elif BARNEY_BACKEND_CUDA
        return (BNContext)createContext_cuda(dataGroupIDs,numGPUs,_gpuIDs);
#else
        throw std::runtime_error
          ("explicitly asked for GPU backend, "
           "but optix support not compiled in");
#endif
      }

      // ------------------------------------------------------------------
      // 3) if user did not specify an explicit GPU list, try to
      // create a GPU backend, and fall back to embree if that doesn't
      // work.
      // ------------------------------------------------------------------

#if BARNEY_BACKEND_OPTIX
      try {
        return (BNContext)createContext_optix(dataGroupIDs,numGPUs,_gpuIDs);
      } catch (std::exception &e) {
        std::cerr << "#barney(warn): could not create optix backend (reason: "
                  << e.what() << ")" << std::endl;
      }
#endif
      
#if BARNEY_BACKEND_CUDA
      try {
        return (BNContext)createContext_cuda(dataGroupIDs,numGPUs,_gpuIDs);
      } catch (std::exception &e) {
        std::cerr << "#barney(warn): could not create cuda backend (reason: "
                  << e.what() << ")" << std::endl;
      }
#endif
      
# if BARNEY_BACKEND_EMBREE
      return (BNContext)createContext_embree(dataGroupIDs);
#endif
      throw std::runtime_error("could not generate _any_ backend?!");
      
#if 0
      // ------------------------------------------------------------------
      // create list of GPUs to use for this rank. if specified by user
      // we use this; otherwise we use GPUs in order, split into groups
      // according to how many ranks there are on this host. Ie, if host
      // has four GPUs the first rank will take 0 and 1; and the second
      // one will take 2 and 3.
      // ------------------------------------------------------------------
      std::vector<int> gpuIDs;
      if (_gpuIDs) {
        for (int i = 0;i < numGPUs;i++)
          gpuIDs.push_back(_gpuIDs[i]);
      }
      else {
        if (numGPUs < 1)
          numGPUs = rtc::Backend::getDeviceCount();
        // hipGetDeviceCount(&numGPUs);
        for (int i = 0;i < numGPUs;i++)
          gpuIDs.push_back(i);
      }
      if (gpuIDs.empty())
        throw std::runtime_error
          ("no devices found!?");

      if (gpuIDs.size() < numDataRanksOnThisContext) {
        std::vector<int> replicatedIDs;
        for (int i = 0;i < numDataRanksOnThisContext;i++)
          replicatedIDs.push_back(gpuIDs[i % gpuIDs.size()]);
        gpuIDs = replicatedIDs;
      }

      return (BNContext)new LocalContext(dataGroupIDs,
                                         gpuIDs);
#endif
    } 
    catch (const std::exception& e) {
      std::cerr << "error creating barney context : " << e.what() << std::endl;
      return 0;
    }
    return 0;
  }

#if BARNEY_MPI
  BARNEY_API
  BNContext bnMPIContextCreate(MPI_Comm _comm,
                               /*! how many data slots this context is to
                                 offer, and which part(s) of the
                                 distributed model data these slot(s)
                                 will hold */
                               const int *dataRanksOnThisContext,
                               int        numDataRanksOnThisContext,
                               /*! which gpu(s) to use for this
                                 process. default is to distribute
                                 node's GPUs equally over all ranks on
                                 that given node */
                               const int *_gpuIDs,
                               int  numGPUs
                               )
  {
    LOG_API_ENTRY;
    if (getenv("BARNEY_FORCE_CPU")) {
	    static int negOne = -1;
	    _gpuIDs = &negOne;
	    numGPUs = 1;
    }
    mpi::Comm world(_comm);
    if (world.size == 1) {
      // std::cout << "#bn: MPIContextInit, but only one rank - using
      // local context" << std::endl;
      return bnContextCreate(dataRanksOnThisContext,
                             numDataRanksOnThisContext == 0
                             ? 1 : numDataRanksOnThisContext,
                             /*! which gpu(s) to use for this
                               process. default is to distribute
                               node's GPUs equally over all ranks on
                               that given node */
                             _gpuIDs,
                             numGPUs);
    }


    // ------------------------------------------------------------------
    // create vector of data groups; if actual specified by user we
    // use those; otherwise we use IDs
    // [0,1,...numModelSlotsOnThisHost)
    // ------------------------------------------------------------------
    assert(/* data groups == 0 is allowed for passive nodes*/
           numDataRanksOnThisContext >= 0);
    std::vector<int> dataGroupIDs;
    int rank;
    MPI_Comm_rank(world, &rank);
    for (int i=0;i<numDataRanksOnThisContext;i++)
      dataGroupIDs.push_back
        (dataRanksOnThisContext
         ? dataRanksOnThisContext[i]
         : rank*numDataRanksOnThisContext+i);

    // check if we're an active worker
    // bool isActiveWorker = !dataGroupIDs.empty();
    // mpi::Comm workers = world.split(isActiveWorker);
    
    // ------------------------------------------------------------------
    // create list of GPUs to use for this rank. if specified by user
    // we use this; otherwise we use GPUs in order, split into groups
    // according to how many ranks there are on this host. Ie, if host
    // has four GPUs the first rank will take 0 and 1; and the second
    // one will take 2 and 3.
    // ------------------------------------------------------------------
    if (_gpuIDs) {
      // gpu IDs _are_ specified by user - use them, or fail
      assert(numGPUs > 0);
      if (
#if BARNEY_BACKEND_EMBREE && !BARNEY_BACKEND_OPTIX
          1
#else
          numGPUs == 1 && _gpuIDs[0] == -1          
#endif
          ) {
        
# if BARNEY_BACKEND_EMBREE
        return (BNContext)createMPIContext_embree(world,
                                                  // workers,
                                                  // isActiveWorker,
                                                  dataGroupIDs);
# else
        throw std::runtime_error
          ("explicitly asked for CPU backend, "
           "but cpu/embree backend not compiled in");
# endif
      }
#if BARNEY_BACKEND_OPTIX
      return (BNContext)createMPIContext_optix(world,
                                               dataGroupIDs,
                                               numGPUs,_gpuIDs);
#else
      throw std::runtime_error("explicitly asked for gpus to use, "
                               "but optix backend not compiled in");
#endif
    }
    PRINT(_gpuIDs);
    throw std::runtime_error("barney mpi-parallel without a list of GPUs is no longer supporteed");
  }


//   BARNEY_API
//   void  bnMPIQueryHardware(BNHardwareInfo *_hardware, MPI_Comm _comm)
//   {
//     LOG_API_ENTRY;

//     assert(_hardware);
//     BNHardwareInfo &hardware = *_hardware;

//     assert(_comm != MPI_COMM_NULL);
//     barney_api::mpi::Comm comm(_comm);

//     hardware.numRanks = comm.size;
//     char hostName[MPI_MAX_PROCESSOR_NAME];
//     memset(hostName,0,MPI_MAX_PROCESSOR_NAME);
//     int hostNameLen = 0;
//     BN_MPI_CALL(Get_processor_name(hostName,&hostNameLen));

//     std::vector<char> recvBuf(MPI_MAX_PROCESSOR_NAME*comm.size);
//     memset(recvBuf.data(),0,recvBuf.size());

//     // ------------------------------------------------------------------
//     // determine which (world) rank lived on which host, and assign
//     // GPUSs
//     // ------------------------------------------------------------------
//     BN_MPI_CALL(Allgather(hostName,
//                           MPI_MAX_PROCESSOR_NAME,MPI_CHAR,
//                           recvBuf.data(),
//                           /* PER rank size */MPI_MAX_PROCESSOR_NAME,MPI_CHAR,
//                           comm.comm));
//     std::vector<std::string>  hostNames;
//     std::map<std::string,int> ranksOnHost;
//     for (int i=0;i<comm.size;i++)  {
//       std::string host_i = recvBuf.data()+i*MPI_MAX_PROCESSOR_NAME;
//       hostNames.push_back(host_i);
//       ranksOnHost[host_i] ++;
//     }

//     hardware.numRanksThisHost = ranksOnHost[hostName];
//     hardware.numHosts         = ranksOnHost.size();

//     // ------------------------------------------------------------------
//     // count how many other ranks are already on this same node
//     // ------------------------------------------------------------------
//     BN_MPI_CALL(Barrier(comm.comm));
//     int localRank = 0;
//     for (int i=0;i<comm.rank;i++)
//       if (hostNames[i] == hostName)
//         localRank++;
//     BN_MPI_CALL(Barrier(comm.comm));
//     hardware.localRank = localRank;
//     hardware.numRanksThisHost = ranksOnHost[hostName];

//     // ------------------------------------------------------------------
//     // assign a GPU to this rank
//     // ------------------------------------------------------------------
//     int numGPUsOnThisHost = 0;
// #if BARNEY_BACKEND_OPTIX
//     hipGetDeviceCount(&numGPUsOnThisHost);
// #endif
//     // hipGetDeviceCount(&numGPUsOnThisHost);
//     // if (numGPUsOnThisHost == 0)
//     //   throw std::runtime_error("no barney-capable devices on this rank!");
//     hardware.numGPUsThisHost = numGPUsOnThisHost;
//     hardware.numGPUsThisRank
//       = comm.allReduceMin(hardware.numGPUsThisHost == 0
//                           ? 0
//                           : std::max(hardware.numGPUsThisHost/
//                                      hardware.numRanksThisHost,
//                                      1));
//   }
  
#endif
} // ::barney_api
